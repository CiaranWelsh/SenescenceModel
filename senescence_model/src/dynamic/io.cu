
/*
 * FLAME GPU v 1.5.X for CUDA 9
 * Copyright University of Sheffield.
 * Original Author: Dr Paul Richmond (user contributions tracked on https://github.com/FLAMEGPU/FLAMEGPU)
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence
 * on www.flamegpu.com website.
 *
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <limits.h>
#include <algorithm>
#include <string>
#include <vector>



#ifdef _WIN32
#define strtok_r strtok_s
#endif

// include header
#include "header.h"

glm::vec3 agent_maximum;
glm::vec3 agent_minimum;

int fpgu_strtol(const char* str){
    return (int)strtol(str, NULL, 0);
}

unsigned int fpgu_strtoul(const char* str){
    return (unsigned int)strtoul(str, NULL, 0);
}

long long int fpgu_strtoll(const char* str){
    return strtoll(str, NULL, 0);
}

unsigned long long int fpgu_strtoull(const char* str){
    return strtoull(str, NULL, 0);
}

double fpgu_strtod(const char* str){
    return strtod(str, NULL);
}

float fgpu_atof(const char* str){
    return (float)atof(str);
}


//templated class function to read array inputs from supported types
template <class T>
void readArrayInput( T (*parseFunc)(const char*), char* buffer, T *array, unsigned int expected_items){
    unsigned int i = 0;
    const char s[2] = ",";
    char * token;
    char * end_str;

    token = strtok_r(buffer, s, &end_str);
    while (token != NULL){
        if (i>=expected_items){
            printf("Error: variable array has too many items, expected %d!\n", expected_items);
            exit(EXIT_FAILURE);
        }
        
        array[i++] = (T)parseFunc(token);
        
        token = strtok_r(NULL, s, &end_str);
    }
    if (i != expected_items){
        printf("Error: variable array has %d items, expected %d!\n", i, expected_items);
        exit(EXIT_FAILURE);
    }
}

//templated class function to read array inputs from supported types
template <class T, class BASE_T, unsigned int D>
void readArrayInputVectorType( BASE_T (*parseFunc)(const char*), char* buffer, T *array, unsigned int expected_items){
    unsigned int i = 0;
    const char s[2] = "|";
    char * token;
    char * end_str;

    token = strtok_r(buffer, s, &end_str);
    while (token != NULL){
        if (i>=expected_items){
            printf("Error: Agent memory array has too many items, expected %d!\n", expected_items);
            exit(EXIT_FAILURE);
        }
        
        //read vector type as an array
        T vec;
        readArrayInput<BASE_T>(parseFunc, token, (BASE_T*) &vec, D);
        array[i++] = vec;
        
        token = strtok_r(NULL, s, &end_str);
    }
    if (i != expected_items){
        printf("Error: Agent memory array has %d items, expected %d!\n", i, expected_items);
        exit(EXIT_FAILURE);
    }
}

void saveIterationData(char* outputpath, int iteration_number, xmachine_memory_A_list* h_As_moving_A, xmachine_memory_A_list* d_As_moving_A, int h_xmachine_memory_A_moving_A_count,xmachine_memory_A_list* h_As_change_direction_A, xmachine_memory_A_list* d_As_change_direction_A, int h_xmachine_memory_A_change_direction_A_count,xmachine_memory_A_list* h_As_get_going_again_A, xmachine_memory_A_list* d_As_get_going_again_A, int h_xmachine_memory_A_get_going_again_A_count,xmachine_memory_B_list* h_Bs_moving_B, xmachine_memory_B_list* d_Bs_moving_B, int h_xmachine_memory_B_moving_B_count,xmachine_memory_B_list* h_Bs_change_direction_B, xmachine_memory_B_list* d_Bs_change_direction_B, int h_xmachine_memory_B_change_direction_B_count,xmachine_memory_B_list* h_Bs_get_going_again_B, xmachine_memory_B_list* d_Bs_get_going_again_B, int h_xmachine_memory_B_get_going_again_B_count)
{
    PROFILE_SCOPED_RANGE("saveIterationData");
	hipError_t cudaStatus;
	
	//Device to host memory transfer
	
	cudaStatus = hipMemcpy( h_As_moving_A, d_As_moving_A, sizeof(xmachine_memory_A_list), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr,"Error Copying A Agent moving_A State Memory from GPU: %s\n", hipGetErrorString(cudaStatus));
		exit(cudaStatus);
	}
	cudaStatus = hipMemcpy( h_As_change_direction_A, d_As_change_direction_A, sizeof(xmachine_memory_A_list), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr,"Error Copying A Agent change_direction_A State Memory from GPU: %s\n", hipGetErrorString(cudaStatus));
		exit(cudaStatus);
	}
	cudaStatus = hipMemcpy( h_As_get_going_again_A, d_As_get_going_again_A, sizeof(xmachine_memory_A_list), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr,"Error Copying A Agent get_going_again_A State Memory from GPU: %s\n", hipGetErrorString(cudaStatus));
		exit(cudaStatus);
	}
	cudaStatus = hipMemcpy( h_Bs_moving_B, d_Bs_moving_B, sizeof(xmachine_memory_B_list), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr,"Error Copying B Agent moving_B State Memory from GPU: %s\n", hipGetErrorString(cudaStatus));
		exit(cudaStatus);
	}
	cudaStatus = hipMemcpy( h_Bs_change_direction_B, d_Bs_change_direction_B, sizeof(xmachine_memory_B_list), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr,"Error Copying B Agent change_direction_B State Memory from GPU: %s\n", hipGetErrorString(cudaStatus));
		exit(cudaStatus);
	}
	cudaStatus = hipMemcpy( h_Bs_get_going_again_B, d_Bs_get_going_again_B, sizeof(xmachine_memory_B_list), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr,"Error Copying B Agent get_going_again_B State Memory from GPU: %s\n", hipGetErrorString(cudaStatus));
		exit(cudaStatus);
	}
	
	/* Pointer to file */
	FILE *file;
	char data[100];

	sprintf(data, "%s%i.xml", outputpath, iteration_number);
	//printf("Writing iteration %i data to %s\n", iteration_number, data);
	file = fopen(data, "w");
    if(file == nullptr){
        printf("Error: Could not open file `%s` for output. Aborting.\n", data);
        exit(EXIT_FAILURE);
    }
    fputs("<states>\n<itno>", file);
    sprintf(data, "%i", iteration_number);
    fputs(data, file);
    fputs("</itno>\n", file);
    fputs("<environment>\n" , file);
    
	fputs("</environment>\n" , file);

	//Write each A agent to xml
	for (int i=0; i<h_xmachine_memory_A_moving_A_count; i++){
		fputs("<xagent>\n" , file);
		fputs("<name>A</name>\n", file);
        
		fputs("<id>", file);
        sprintf(data, "%d", h_As_moving_A->id[i]);
		fputs(data, file);
		fputs("</id>\n", file);
        
		fputs("<x>", file);
        sprintf(data, "%f", h_As_moving_A->x[i]);
		fputs(data, file);
		fputs("</x>\n", file);
        
		fputs("<y>", file);
        sprintf(data, "%f", h_As_moving_A->y[i]);
		fputs(data, file);
		fputs("</y>\n", file);
        
		fputs("<z>", file);
        sprintf(data, "%f", h_As_moving_A->z[i]);
		fputs(data, file);
		fputs("</z>\n", file);
        
		fputs("<fx>", file);
        sprintf(data, "%f", h_As_moving_A->fx[i]);
		fputs(data, file);
		fputs("</fx>\n", file);
        
		fputs("<fy>", file);
        sprintf(data, "%f", h_As_moving_A->fy[i]);
		fputs(data, file);
		fputs("</fy>\n", file);
        
		fputs("<fz>", file);
        sprintf(data, "%f", h_As_moving_A->fz[i]);
		fputs(data, file);
		fputs("</fz>\n", file);
        
		fputs("<colour>", file);
        sprintf(data, "%d", h_As_moving_A->colour[i]);
		fputs(data, file);
		fputs("</colour>\n", file);
        
		fputs("</xagent>\n", file);
	}
	//Write each A agent to xml
	for (int i=0; i<h_xmachine_memory_A_change_direction_A_count; i++){
		fputs("<xagent>\n" , file);
		fputs("<name>A</name>\n", file);
        
		fputs("<id>", file);
        sprintf(data, "%d", h_As_change_direction_A->id[i]);
		fputs(data, file);
		fputs("</id>\n", file);
        
		fputs("<x>", file);
        sprintf(data, "%f", h_As_change_direction_A->x[i]);
		fputs(data, file);
		fputs("</x>\n", file);
        
		fputs("<y>", file);
        sprintf(data, "%f", h_As_change_direction_A->y[i]);
		fputs(data, file);
		fputs("</y>\n", file);
        
		fputs("<z>", file);
        sprintf(data, "%f", h_As_change_direction_A->z[i]);
		fputs(data, file);
		fputs("</z>\n", file);
        
		fputs("<fx>", file);
        sprintf(data, "%f", h_As_change_direction_A->fx[i]);
		fputs(data, file);
		fputs("</fx>\n", file);
        
		fputs("<fy>", file);
        sprintf(data, "%f", h_As_change_direction_A->fy[i]);
		fputs(data, file);
		fputs("</fy>\n", file);
        
		fputs("<fz>", file);
        sprintf(data, "%f", h_As_change_direction_A->fz[i]);
		fputs(data, file);
		fputs("</fz>\n", file);
        
		fputs("<colour>", file);
        sprintf(data, "%d", h_As_change_direction_A->colour[i]);
		fputs(data, file);
		fputs("</colour>\n", file);
        
		fputs("</xagent>\n", file);
	}
	//Write each A agent to xml
	for (int i=0; i<h_xmachine_memory_A_get_going_again_A_count; i++){
		fputs("<xagent>\n" , file);
		fputs("<name>A</name>\n", file);
        
		fputs("<id>", file);
        sprintf(data, "%d", h_As_get_going_again_A->id[i]);
		fputs(data, file);
		fputs("</id>\n", file);
        
		fputs("<x>", file);
        sprintf(data, "%f", h_As_get_going_again_A->x[i]);
		fputs(data, file);
		fputs("</x>\n", file);
        
		fputs("<y>", file);
        sprintf(data, "%f", h_As_get_going_again_A->y[i]);
		fputs(data, file);
		fputs("</y>\n", file);
        
		fputs("<z>", file);
        sprintf(data, "%f", h_As_get_going_again_A->z[i]);
		fputs(data, file);
		fputs("</z>\n", file);
        
		fputs("<fx>", file);
        sprintf(data, "%f", h_As_get_going_again_A->fx[i]);
		fputs(data, file);
		fputs("</fx>\n", file);
        
		fputs("<fy>", file);
        sprintf(data, "%f", h_As_get_going_again_A->fy[i]);
		fputs(data, file);
		fputs("</fy>\n", file);
        
		fputs("<fz>", file);
        sprintf(data, "%f", h_As_get_going_again_A->fz[i]);
		fputs(data, file);
		fputs("</fz>\n", file);
        
		fputs("<colour>", file);
        sprintf(data, "%d", h_As_get_going_again_A->colour[i]);
		fputs(data, file);
		fputs("</colour>\n", file);
        
		fputs("</xagent>\n", file);
	}
	//Write each B agent to xml
	for (int i=0; i<h_xmachine_memory_B_moving_B_count; i++){
		fputs("<xagent>\n" , file);
		fputs("<name>B</name>\n", file);
        
		fputs("<id>", file);
        sprintf(data, "%d", h_Bs_moving_B->id[i]);
		fputs(data, file);
		fputs("</id>\n", file);
        
		fputs("<x>", file);
        sprintf(data, "%f", h_Bs_moving_B->x[i]);
		fputs(data, file);
		fputs("</x>\n", file);
        
		fputs("<y>", file);
        sprintf(data, "%f", h_Bs_moving_B->y[i]);
		fputs(data, file);
		fputs("</y>\n", file);
        
		fputs("<z>", file);
        sprintf(data, "%f", h_Bs_moving_B->z[i]);
		fputs(data, file);
		fputs("</z>\n", file);
        
		fputs("<fx>", file);
        sprintf(data, "%f", h_Bs_moving_B->fx[i]);
		fputs(data, file);
		fputs("</fx>\n", file);
        
		fputs("<fy>", file);
        sprintf(data, "%f", h_Bs_moving_B->fy[i]);
		fputs(data, file);
		fputs("</fy>\n", file);
        
		fputs("<fz>", file);
        sprintf(data, "%f", h_Bs_moving_B->fz[i]);
		fputs(data, file);
		fputs("</fz>\n", file);
        
		fputs("<colour>", file);
        sprintf(data, "%d", h_Bs_moving_B->colour[i]);
		fputs(data, file);
		fputs("</colour>\n", file);
        
		fputs("</xagent>\n", file);
	}
	//Write each B agent to xml
	for (int i=0; i<h_xmachine_memory_B_change_direction_B_count; i++){
		fputs("<xagent>\n" , file);
		fputs("<name>B</name>\n", file);
        
		fputs("<id>", file);
        sprintf(data, "%d", h_Bs_change_direction_B->id[i]);
		fputs(data, file);
		fputs("</id>\n", file);
        
		fputs("<x>", file);
        sprintf(data, "%f", h_Bs_change_direction_B->x[i]);
		fputs(data, file);
		fputs("</x>\n", file);
        
		fputs("<y>", file);
        sprintf(data, "%f", h_Bs_change_direction_B->y[i]);
		fputs(data, file);
		fputs("</y>\n", file);
        
		fputs("<z>", file);
        sprintf(data, "%f", h_Bs_change_direction_B->z[i]);
		fputs(data, file);
		fputs("</z>\n", file);
        
		fputs("<fx>", file);
        sprintf(data, "%f", h_Bs_change_direction_B->fx[i]);
		fputs(data, file);
		fputs("</fx>\n", file);
        
		fputs("<fy>", file);
        sprintf(data, "%f", h_Bs_change_direction_B->fy[i]);
		fputs(data, file);
		fputs("</fy>\n", file);
        
		fputs("<fz>", file);
        sprintf(data, "%f", h_Bs_change_direction_B->fz[i]);
		fputs(data, file);
		fputs("</fz>\n", file);
        
		fputs("<colour>", file);
        sprintf(data, "%d", h_Bs_change_direction_B->colour[i]);
		fputs(data, file);
		fputs("</colour>\n", file);
        
		fputs("</xagent>\n", file);
	}
	//Write each B agent to xml
	for (int i=0; i<h_xmachine_memory_B_get_going_again_B_count; i++){
		fputs("<xagent>\n" , file);
		fputs("<name>B</name>\n", file);
        
		fputs("<id>", file);
        sprintf(data, "%d", h_Bs_get_going_again_B->id[i]);
		fputs(data, file);
		fputs("</id>\n", file);
        
		fputs("<x>", file);
        sprintf(data, "%f", h_Bs_get_going_again_B->x[i]);
		fputs(data, file);
		fputs("</x>\n", file);
        
		fputs("<y>", file);
        sprintf(data, "%f", h_Bs_get_going_again_B->y[i]);
		fputs(data, file);
		fputs("</y>\n", file);
        
		fputs("<z>", file);
        sprintf(data, "%f", h_Bs_get_going_again_B->z[i]);
		fputs(data, file);
		fputs("</z>\n", file);
        
		fputs("<fx>", file);
        sprintf(data, "%f", h_Bs_get_going_again_B->fx[i]);
		fputs(data, file);
		fputs("</fx>\n", file);
        
		fputs("<fy>", file);
        sprintf(data, "%f", h_Bs_get_going_again_B->fy[i]);
		fputs(data, file);
		fputs("</fy>\n", file);
        
		fputs("<fz>", file);
        sprintf(data, "%f", h_Bs_get_going_again_B->fz[i]);
		fputs(data, file);
		fputs("</fz>\n", file);
        
		fputs("<colour>", file);
        sprintf(data, "%d", h_Bs_get_going_again_B->colour[i]);
		fputs(data, file);
		fputs("</colour>\n", file);
        
		fputs("</xagent>\n", file);
	}
	
	

	fputs("</states>\n" , file);
	
	/* Close the file */
	fclose(file);

}

void readInitialStates(char* inputpath, xmachine_memory_A_list* h_As, int* h_xmachine_memory_A_count,xmachine_memory_B_list* h_Bs, int* h_xmachine_memory_B_count)
{
    PROFILE_SCOPED_RANGE("readInitialStates");

	int temp = 0;
	int* itno = &temp;

	/* Pointer to file */
	FILE *file;
	/* Char and char buffer for reading file to */
	char c = ' ';
	const int bufferSize = 10000;
	char buffer[bufferSize];
	char agentname[1000];

	/* Pointer to x-memory for initial state data */
	/*xmachine * current_xmachine;*/
	/* Variables for checking tags */
	int reading, i;
	int in_tag, in_itno, in_xagent, in_name, in_comment;
    int in_A_id;
    int in_A_x;
    int in_A_y;
    int in_A_z;
    int in_A_fx;
    int in_A_fy;
    int in_A_fz;
    int in_A_colour;
    int in_B_id;
    int in_B_x;
    int in_B_y;
    int in_B_z;
    int in_B_fx;
    int in_B_fy;
    int in_B_fz;
    int in_B_colour;
    
    /* tags for environment global variables */
    int in_env;
	/* set agent count to zero */
	*h_xmachine_memory_A_count = 0;
	*h_xmachine_memory_B_count = 0;
	
	/* Variables for initial state data */
	int A_id;
	float A_x;
	float A_y;
	float A_z;
	float A_fx;
	float A_fy;
	float A_fz;
	int A_colour;
	int B_id;
	float B_x;
	float B_y;
	float B_z;
	float B_fx;
	float B_fy;
	float B_fz;
	int B_colour;

    /* Variables for environment variables */
    


	/* Initialise variables */
    agent_maximum.x = 0;
    agent_maximum.y = 0;
    agent_maximum.z = 0;
    agent_minimum.x = 0;
    agent_minimum.y = 0;
    agent_minimum.z = 0;
	reading = 1;
    in_comment = 0;
	in_tag = 0;
	in_itno = 0;
    in_env = 0;
    in_xagent = 0;
	in_name = 0;
	in_A_id = 0;
	in_A_x = 0;
	in_A_y = 0;
	in_A_z = 0;
	in_A_fx = 0;
	in_A_fy = 0;
	in_A_fz = 0;
	in_A_colour = 0;
	in_B_id = 0;
	in_B_x = 0;
	in_B_y = 0;
	in_B_z = 0;
	in_B_fx = 0;
	in_B_fy = 0;
	in_B_fz = 0;
	in_B_colour = 0;
	//set all A values to 0
	//If this is not done then it will cause errors in emu mode where undefined memory is not 0
	for (int k=0; k<xmachine_memory_A_MAX; k++)
	{	
		h_As->id[k] = 0;
		h_As->x[k] = 0;
		h_As->y[k] = 0;
		h_As->z[k] = 0;
		h_As->fx[k] = 0;
		h_As->fy[k] = 0;
		h_As->fz[k] = 0;
		h_As->colour[k] = 0;
	}
	
	//set all B values to 0
	//If this is not done then it will cause errors in emu mode where undefined memory is not 0
	for (int k=0; k<xmachine_memory_B_MAX; k++)
	{	
		h_Bs->id[k] = 0;
		h_Bs->x[k] = 0;
		h_Bs->y[k] = 0;
		h_Bs->z[k] = 0;
		h_Bs->fx[k] = 0;
		h_Bs->fy[k] = 0;
		h_Bs->fz[k] = 0;
		h_Bs->colour[k] = 0;
	}
	

	/* Default variables for memory */
    A_id = 0;
    A_x = 0;
    A_y = 0;
    A_z = 0;
    A_fx = 0;
    A_fy = 0;
    A_fz = 0;
    A_colour = 0;
    B_id = 0;
    B_x = 0;
    B_y = 0;
    B_z = 0;
    B_fx = 0;
    B_fy = 0;
    B_fz = 0;
    B_colour = 0;

    /* Default variables for environment variables */
    
    
    // If no input path was specified, issue a message and return.
    if(inputpath[0] == '\0'){
        printf("No initial states file specified. Using default values.\n");
        return;
    }
    
    // Otherwise an input path was specified, and we have previously checked that it is (was) not a directory. 
    
	// Attempt to open the non directory path as read only.
	file = fopen(inputpath, "r");
    
    // If the file could not be opened, issue a message and return.
    if(file == nullptr)
    {
      printf("Could not open input file %s. Continuing with default values\n", inputpath);
      return;
    }
    // Otherwise we can iterate the file until the end of XML is reached.
    size_t bytesRead = 0;
    i = 0;
	while(reading==1)
	{
        // If I exceeds our buffer size we must abort
        if(i >= bufferSize){
            fprintf(stderr, "Error: XML Parsing failed Tag name or content too long (> %d characters)\n", bufferSize);
            exit(EXIT_FAILURE);
        }

		/* Get the next char from the file */
		c = (char)fgetc(file);

        // Check if we reached the end of the file.
        if(c == EOF){
            // Break out of the loop. This allows for empty files(which may or may not be)
            break;
        }
        // Increment byte counter.
        bytesRead++;

        /*If in a  comment, look for the end of a comment */
        if(in_comment){

            /* Look for an end tag following two (or more) hyphens.
               To support very long comments, we use the minimal amount of buffer we can. 
               If we see a hyphen, store it and increment i (but don't increment i)
               If we see a > check if we have a correct terminating comment
               If we see any other characters, reset i.
            */

            if(c == '-'){
                buffer[i] = c;
                i++;
            } else if(c == '>' && i >= 2){
                in_comment = 0;
                i = 0;
            } else {
                i = 0;
            }

            /*// If we see the end tag, check the preceding two characters for a close comment, if enough characters have been read for -->
            if(c == '>' && i >= 2 && buffer[i-1] == '-' && buffer[i-2] == '-'){
                in_comment = 0;
                buffer[0] = 0;
                i = 0;
            } else {
                // Otherwise just store it in the buffer so we can keep checking for close tags
                buffer[i] = c;
                i++;
            }*/
        }
		/* If the end of a tag */
		else if(c == '>')
		{
			/* Place 0 at end of buffer to make chars a string */
			buffer[i] = 0;

			if(strcmp(buffer, "states") == 0) reading = 1;
			if(strcmp(buffer, "/states") == 0) reading = 0;
			if(strcmp(buffer, "itno") == 0) in_itno = 1;
			if(strcmp(buffer, "/itno") == 0) in_itno = 0;
            if(strcmp(buffer, "environment") == 0) in_env = 1;
            if(strcmp(buffer, "/environment") == 0) in_env = 0;
			if(strcmp(buffer, "name") == 0) in_name = 1;
			if(strcmp(buffer, "/name") == 0) in_name = 0;
            if(strcmp(buffer, "xagent") == 0) in_xagent = 1;
			if(strcmp(buffer, "/xagent") == 0)
			{
				if(strcmp(agentname, "A") == 0)
				{
					if (*h_xmachine_memory_A_count > xmachine_memory_A_MAX){
						printf("ERROR: MAX Buffer size (%i) for agent A exceeded whilst reading data\n", xmachine_memory_A_MAX);
						// Close the file and stop reading
						fclose(file);
						exit(EXIT_FAILURE);
					}
                    
					h_As->id[*h_xmachine_memory_A_count] = A_id;
					h_As->x[*h_xmachine_memory_A_count] = A_x;//Check maximum x value
                    if(agent_maximum.x < A_x)
                        agent_maximum.x = (float)A_x;
                    //Check minimum x value
                    if(agent_minimum.x > A_x)
                        agent_minimum.x = (float)A_x;
                    
					h_As->y[*h_xmachine_memory_A_count] = A_y;//Check maximum y value
                    if(agent_maximum.y < A_y)
                        agent_maximum.y = (float)A_y;
                    //Check minimum y value
                    if(agent_minimum.y > A_y)
                        agent_minimum.y = (float)A_y;
                    
					h_As->z[*h_xmachine_memory_A_count] = A_z;//Check maximum z value
                    if(agent_maximum.z < A_z)
                        agent_maximum.z = (float)A_z;
                    //Check minimum z value
                    if(agent_minimum.z > A_z)
                        agent_minimum.z = (float)A_z;
                    
					h_As->fx[*h_xmachine_memory_A_count] = A_fx;
					h_As->fy[*h_xmachine_memory_A_count] = A_fy;
					h_As->fz[*h_xmachine_memory_A_count] = A_fz;
					h_As->colour[*h_xmachine_memory_A_count] = A_colour;
					(*h_xmachine_memory_A_count) ++;	
				}
				else if(strcmp(agentname, "B") == 0)
				{
					if (*h_xmachine_memory_B_count > xmachine_memory_B_MAX){
						printf("ERROR: MAX Buffer size (%i) for agent B exceeded whilst reading data\n", xmachine_memory_B_MAX);
						// Close the file and stop reading
						fclose(file);
						exit(EXIT_FAILURE);
					}
                    
					h_Bs->id[*h_xmachine_memory_B_count] = B_id;
					h_Bs->x[*h_xmachine_memory_B_count] = B_x;//Check maximum x value
                    if(agent_maximum.x < B_x)
                        agent_maximum.x = (float)B_x;
                    //Check minimum x value
                    if(agent_minimum.x > B_x)
                        agent_minimum.x = (float)B_x;
                    
					h_Bs->y[*h_xmachine_memory_B_count] = B_y;//Check maximum y value
                    if(agent_maximum.y < B_y)
                        agent_maximum.y = (float)B_y;
                    //Check minimum y value
                    if(agent_minimum.y > B_y)
                        agent_minimum.y = (float)B_y;
                    
					h_Bs->z[*h_xmachine_memory_B_count] = B_z;//Check maximum z value
                    if(agent_maximum.z < B_z)
                        agent_maximum.z = (float)B_z;
                    //Check minimum z value
                    if(agent_minimum.z > B_z)
                        agent_minimum.z = (float)B_z;
                    
					h_Bs->fx[*h_xmachine_memory_B_count] = B_fx;
					h_Bs->fy[*h_xmachine_memory_B_count] = B_fy;
					h_Bs->fz[*h_xmachine_memory_B_count] = B_fz;
					h_Bs->colour[*h_xmachine_memory_B_count] = B_colour;
					(*h_xmachine_memory_B_count) ++;	
				}
				else
				{
					printf("Warning: agent name undefined - '%s'\n", agentname);
				}



				/* Reset xagent variables */
                A_id = 0;
                A_x = 0;
                A_y = 0;
                A_z = 0;
                A_fx = 0;
                A_fy = 0;
                A_fz = 0;
                A_colour = 0;
                B_id = 0;
                B_x = 0;
                B_y = 0;
                B_z = 0;
                B_fx = 0;
                B_fy = 0;
                B_fz = 0;
                B_colour = 0;
                
                in_xagent = 0;
			}
			if(strcmp(buffer, "id") == 0) in_A_id = 1;
			if(strcmp(buffer, "/id") == 0) in_A_id = 0;
			if(strcmp(buffer, "x") == 0) in_A_x = 1;
			if(strcmp(buffer, "/x") == 0) in_A_x = 0;
			if(strcmp(buffer, "y") == 0) in_A_y = 1;
			if(strcmp(buffer, "/y") == 0) in_A_y = 0;
			if(strcmp(buffer, "z") == 0) in_A_z = 1;
			if(strcmp(buffer, "/z") == 0) in_A_z = 0;
			if(strcmp(buffer, "fx") == 0) in_A_fx = 1;
			if(strcmp(buffer, "/fx") == 0) in_A_fx = 0;
			if(strcmp(buffer, "fy") == 0) in_A_fy = 1;
			if(strcmp(buffer, "/fy") == 0) in_A_fy = 0;
			if(strcmp(buffer, "fz") == 0) in_A_fz = 1;
			if(strcmp(buffer, "/fz") == 0) in_A_fz = 0;
			if(strcmp(buffer, "colour") == 0) in_A_colour = 1;
			if(strcmp(buffer, "/colour") == 0) in_A_colour = 0;
			if(strcmp(buffer, "id") == 0) in_B_id = 1;
			if(strcmp(buffer, "/id") == 0) in_B_id = 0;
			if(strcmp(buffer, "x") == 0) in_B_x = 1;
			if(strcmp(buffer, "/x") == 0) in_B_x = 0;
			if(strcmp(buffer, "y") == 0) in_B_y = 1;
			if(strcmp(buffer, "/y") == 0) in_B_y = 0;
			if(strcmp(buffer, "z") == 0) in_B_z = 1;
			if(strcmp(buffer, "/z") == 0) in_B_z = 0;
			if(strcmp(buffer, "fx") == 0) in_B_fx = 1;
			if(strcmp(buffer, "/fx") == 0) in_B_fx = 0;
			if(strcmp(buffer, "fy") == 0) in_B_fy = 1;
			if(strcmp(buffer, "/fy") == 0) in_B_fy = 0;
			if(strcmp(buffer, "fz") == 0) in_B_fz = 1;
			if(strcmp(buffer, "/fz") == 0) in_B_fz = 0;
			if(strcmp(buffer, "colour") == 0) in_B_colour = 1;
			if(strcmp(buffer, "/colour") == 0) in_B_colour = 0;
			
            /* environment variables */
            

			/* End of tag and reset buffer */
			in_tag = 0;
			i = 0;
		}
		/* If start of tag */
		else if(c == '<')
		{
			/* Place /0 at end of buffer to end numbers */
			buffer[i] = 0;
			/* Flag in tag */
			in_tag = 1;

			if(in_itno) *itno = atoi(buffer);
			if(in_name) strcpy(agentname, buffer);
			else if (in_xagent)
			{
				if(in_A_id){
                    A_id = (int) fpgu_strtol(buffer); 
                }
				if(in_A_x){
                    A_x = (float) fgpu_atof(buffer); 
                }
				if(in_A_y){
                    A_y = (float) fgpu_atof(buffer); 
                }
				if(in_A_z){
                    A_z = (float) fgpu_atof(buffer); 
                }
				if(in_A_fx){
                    A_fx = (float) fgpu_atof(buffer); 
                }
				if(in_A_fy){
                    A_fy = (float) fgpu_atof(buffer); 
                }
				if(in_A_fz){
                    A_fz = (float) fgpu_atof(buffer); 
                }
				if(in_A_colour){
                    A_colour = (int) fpgu_strtol(buffer); 
                }
				if(in_B_id){
                    B_id = (int) fpgu_strtol(buffer); 
                }
				if(in_B_x){
                    B_x = (float) fgpu_atof(buffer); 
                }
				if(in_B_y){
                    B_y = (float) fgpu_atof(buffer); 
                }
				if(in_B_z){
                    B_z = (float) fgpu_atof(buffer); 
                }
				if(in_B_fx){
                    B_fx = (float) fgpu_atof(buffer); 
                }
				if(in_B_fy){
                    B_fy = (float) fgpu_atof(buffer); 
                }
				if(in_B_fz){
                    B_fz = (float) fgpu_atof(buffer); 
                }
				if(in_B_colour){
                    B_colour = (int) fpgu_strtol(buffer); 
                }
				
            }
            else if (in_env){
            
            }
		/* Reset buffer */
			i = 0;
		}
		/* If in tag put read char into buffer */
		else if(in_tag)
		{
            // Check if we are a comment, when we are in a tag and buffer[0:2] == "!--"
            if(i == 2 && c == '-' && buffer[1] == '-' && buffer[0] == '!'){
                in_comment = 1;
                // Reset the buffer and i.
                buffer[0] = 0;
                i = 0;
            }

            // Store the character and increment the counter
            buffer[i] = c;
            i++;

		}
		/* If in data read char into buffer */
		else
		{
			buffer[i] = c;
			i++;
		}
	}
    // If no bytes were read, raise a warning.
    if(bytesRead == 0){
        fprintf(stdout, "Warning: %s is an empty file\n", inputpath);
        fflush(stdout);
    }

    // If the in_comment flag is still marked, issue a warning.
    if(in_comment){
        fprintf(stdout, "Warning: Un-terminated comment in %s\n", inputpath);
        fflush(stdout);
    }    

	/* Close the file */
	fclose(file);
}

glm::vec3 getMaximumBounds(){
    return agent_maximum;
}

glm::vec3 getMinimumBounds(){
    return agent_minimum;
}


/* Methods to load static networks from disk */
