#include "hip/hip_runtime.h"

/*
 * FLAME GPU v 1.5.X for CUDA 9
 * Copyright University of Sheffield.
 * Original Author: Dr Paul Richmond (user contributions tracked on https://github.com/FLAMEGPU/FLAMEGPU)
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence
 * on www.flamegpu.com website.
 *
 */


#ifndef _FLAMEGPU_KERNELS_H_
#define _FLAMEGPU_KERNELS_H_

#include "header.h"


/* Agent count constants */

__constant__ int d_xmachine_memory_TissueBlock_count;

__constant__ int d_xmachine_memory_Fibroblast_count;

/* Agent state count constants */

__constant__ int d_xmachine_memory_TissueBlock_default_count;

__constant__ int d_xmachine_memory_Fibroblast_Quiescent_count;

__constant__ int d_xmachine_memory_Fibroblast_EarlySenescent_count;

__constant__ int d_xmachine_memory_Fibroblast_Senescent_count;

__constant__ int d_xmachine_memory_Fibroblast_Proliferating_count;

__constant__ int d_xmachine_memory_Fibroblast_Repair_count;


/* Message constants */

/* fibroblast_damage_report Message variables */
/* Non partitioned, spatial partitioned and on-graph partitioned message variables  */
__constant__ int d_message_fibroblast_damage_report_count;         /**< message list counter*/
__constant__ int d_message_fibroblast_damage_report_output_type;   /**< message output type (single or optional)*/
//Spatial Partitioning Variables
__constant__ glm::vec3 d_message_fibroblast_damage_report_min_bounds;           /**< min bounds (x,y,z) of partitioning environment */
__constant__ glm::vec3 d_message_fibroblast_damage_report_max_bounds;           /**< max bounds (x,y,z) of partitioning environment */
__constant__ glm::ivec3 d_message_fibroblast_damage_report_partitionDim;           /**< partition dimensions (x,y,z) of partitioning environment */
__constant__ float d_message_fibroblast_damage_report_radius;                 /**< partition radius (used to determin the size of the partitions) */

/* tissue_damage_report Message variables */
/* Non partitioned, spatial partitioned and on-graph partitioned message variables  */
__constant__ int d_message_tissue_damage_report_count;         /**< message list counter*/
__constant__ int d_message_tissue_damage_report_output_type;   /**< message output type (single or optional)*/
//Spatial Partitioning Variables
__constant__ glm::vec3 d_message_tissue_damage_report_min_bounds;           /**< min bounds (x,y,z) of partitioning environment */
__constant__ glm::vec3 d_message_tissue_damage_report_max_bounds;           /**< max bounds (x,y,z) of partitioning environment */
__constant__ glm::ivec3 d_message_tissue_damage_report_partitionDim;           /**< partition dimensions (x,y,z) of partitioning environment */
__constant__ float d_message_tissue_damage_report_radius;                 /**< partition radius (used to determin the size of the partitions) */

/* doublings Message variables */
/* Non partitioned, spatial partitioned and on-graph partitioned message variables  */
__constant__ int d_message_doublings_count;         /**< message list counter*/
__constant__ int d_message_doublings_output_type;   /**< message output type (single or optional)*/

/* count Message variables */
/* Non partitioned, spatial partitioned and on-graph partitioned message variables  */
__constant__ int d_message_count_count;         /**< message list counter*/
__constant__ int d_message_count_output_type;   /**< message output type (single or optional)*/

/* quiescent_location_report Message variables */
/* Non partitioned, spatial partitioned and on-graph partitioned message variables  */
__constant__ int d_message_quiescent_location_report_count;         /**< message list counter*/
__constant__ int d_message_quiescent_location_report_output_type;   /**< message output type (single or optional)*/
//Spatial Partitioning Variables
__constant__ glm::vec3 d_message_quiescent_location_report_min_bounds;           /**< min bounds (x,y,z) of partitioning environment */
__constant__ glm::vec3 d_message_quiescent_location_report_max_bounds;           /**< max bounds (x,y,z) of partitioning environment */
__constant__ glm::ivec3 d_message_quiescent_location_report_partitionDim;           /**< partition dimensions (x,y,z) of partitioning environment */
__constant__ float d_message_quiescent_location_report_radius;                 /**< partition radius (used to determin the size of the partitions) */

/* senescent_location_report Message variables */
/* Non partitioned, spatial partitioned and on-graph partitioned message variables  */
__constant__ int d_message_senescent_location_report_count;         /**< message list counter*/
__constant__ int d_message_senescent_location_report_output_type;   /**< message output type (single or optional)*/
//Spatial Partitioning Variables
__constant__ glm::vec3 d_message_senescent_location_report_min_bounds;           /**< min bounds (x,y,z) of partitioning environment */
__constant__ glm::vec3 d_message_senescent_location_report_max_bounds;           /**< max bounds (x,y,z) of partitioning environment */
__constant__ glm::ivec3 d_message_senescent_location_report_partitionDim;           /**< partition dimensions (x,y,z) of partitioning environment */
__constant__ float d_message_senescent_location_report_radius;                 /**< partition radius (used to determin the size of the partitions) */

	

/* Graph Constants */


/* Graph device array pointer(s) */


/* Graph host array pointer(s) */

    
//include each function file

#include "functions.c"
    
/* Texture bindings */
/* fibroblast_damage_report Message Bindings */texture<int, 1, hipReadModeElementType> tex_xmachine_message_fibroblast_damage_report_id;
__constant__ int d_tex_xmachine_message_fibroblast_damage_report_id_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_fibroblast_damage_report_x;
__constant__ int d_tex_xmachine_message_fibroblast_damage_report_x_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_fibroblast_damage_report_y;
__constant__ int d_tex_xmachine_message_fibroblast_damage_report_y_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_fibroblast_damage_report_z;
__constant__ int d_tex_xmachine_message_fibroblast_damage_report_z_offset;texture<int, 1, hipReadModeElementType> tex_xmachine_message_fibroblast_damage_report_damage;
__constant__ int d_tex_xmachine_message_fibroblast_damage_report_damage_offset;
texture<int, 1, hipReadModeElementType> tex_xmachine_message_fibroblast_damage_report_pbm_start;
__constant__ int d_tex_xmachine_message_fibroblast_damage_report_pbm_start_offset;
texture<int, 1, hipReadModeElementType> tex_xmachine_message_fibroblast_damage_report_pbm_end_or_count;
__constant__ int d_tex_xmachine_message_fibroblast_damage_report_pbm_end_or_count_offset;


/* tissue_damage_report Message Bindings */texture<int, 1, hipReadModeElementType> tex_xmachine_message_tissue_damage_report_id;
__constant__ int d_tex_xmachine_message_tissue_damage_report_id_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_tissue_damage_report_x;
__constant__ int d_tex_xmachine_message_tissue_damage_report_x_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_tissue_damage_report_y;
__constant__ int d_tex_xmachine_message_tissue_damage_report_y_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_tissue_damage_report_z;
__constant__ int d_tex_xmachine_message_tissue_damage_report_z_offset;texture<int, 1, hipReadModeElementType> tex_xmachine_message_tissue_damage_report_damage;
__constant__ int d_tex_xmachine_message_tissue_damage_report_damage_offset;
texture<int, 1, hipReadModeElementType> tex_xmachine_message_tissue_damage_report_pbm_start;
__constant__ int d_tex_xmachine_message_tissue_damage_report_pbm_start_offset;
texture<int, 1, hipReadModeElementType> tex_xmachine_message_tissue_damage_report_pbm_end_or_count;
__constant__ int d_tex_xmachine_message_tissue_damage_report_pbm_end_or_count_offset;




/* quiescent_location_report Message Bindings */texture<int, 1, hipReadModeElementType> tex_xmachine_message_quiescent_location_report_id;
__constant__ int d_tex_xmachine_message_quiescent_location_report_id_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_quiescent_location_report_x;
__constant__ int d_tex_xmachine_message_quiescent_location_report_x_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_quiescent_location_report_y;
__constant__ int d_tex_xmachine_message_quiescent_location_report_y_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_quiescent_location_report_z;
__constant__ int d_tex_xmachine_message_quiescent_location_report_z_offset;
texture<int, 1, hipReadModeElementType> tex_xmachine_message_quiescent_location_report_pbm_start;
__constant__ int d_tex_xmachine_message_quiescent_location_report_pbm_start_offset;
texture<int, 1, hipReadModeElementType> tex_xmachine_message_quiescent_location_report_pbm_end_or_count;
__constant__ int d_tex_xmachine_message_quiescent_location_report_pbm_end_or_count_offset;


/* senescent_location_report Message Bindings */texture<int, 1, hipReadModeElementType> tex_xmachine_message_senescent_location_report_id;
__constant__ int d_tex_xmachine_message_senescent_location_report_id_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_senescent_location_report_x;
__constant__ int d_tex_xmachine_message_senescent_location_report_x_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_senescent_location_report_y;
__constant__ int d_tex_xmachine_message_senescent_location_report_y_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_senescent_location_report_z;
__constant__ int d_tex_xmachine_message_senescent_location_report_z_offset;
texture<int, 1, hipReadModeElementType> tex_xmachine_message_senescent_location_report_pbm_start;
__constant__ int d_tex_xmachine_message_senescent_location_report_pbm_start_offset;
texture<int, 1, hipReadModeElementType> tex_xmachine_message_senescent_location_report_pbm_end_or_count;
__constant__ int d_tex_xmachine_message_senescent_location_report_pbm_end_or_count_offset;


    
#define WRAP(x,m) (((x)<m)?(x):(x%m)) /**< Simple wrap */
#define sWRAP(x,m) (((x)<m)?(((x)<0)?(m+(x)):(x)):(m-(x))) /**<signed integer wrap (no modulus) for negatives where 2m > |x| > m */

//PADDING WILL ONLY AVOID SM CONFLICTS FOR 32BIT
//SM_OFFSET REQUIRED AS FERMI STARTS INDEXING MEMORY FROM LOCATION 0 (i.e. NULL)??
__constant__ int d_SM_START;
__constant__ int d_PADDING;

//SM addressing macro to avoid conflicts (32 bit only)
#define SHARE_INDEX(i, s) ((((s) + d_PADDING)* (i))+d_SM_START) /**<offset struct size by padding to avoid bank conflicts */

//if doubel support is needed then define the following function which requires sm_13 or later
#ifdef _DOUBLE_SUPPORT_REQUIRED_
__inline__ __device__ double tex1DfetchDouble(texture<int2, 1, hipReadModeElementType> tex, int i)
{
	int2 v = tex1Dfetch(tex, i);
  //IF YOU HAVE AN ERROR HERE THEN YOU ARE USING DOUBLE VALUES IN AGENT MEMORY AND NOT COMPILING FOR DOUBLE SUPPORTED HARDWARE
  //To compile for double supported hardware change the CUDA Build rule property "Use sm_13 Architecture (double support)" on the CUDA-Specific Propert Page of the CUDA Build Rule for simulation.cu
	return __hiloint2double(v.y, v.x);
}
#endif

/* Helper functions */
/** next_cell
 * Function used for finding the next cell when using spatial partitioning
 * Upddates the relative cell variable which can have value of -1, 0 or +1
 * @param relative_cell pointer to the relative cell position
 * @return boolean if there is a next cell. True unless relative_Cell value was 1,1,1
 */
__device__ bool next_cell3D(glm::ivec3* relative_cell)
{
	if (relative_cell->x < 1)
	{
		relative_cell->x++;
		return true;
	}
	relative_cell->x = -1;

	if (relative_cell->y < 1)
	{
		relative_cell->y++;
		return true;
	}
	relative_cell->y = -1;
	
	if (relative_cell->z < 1)
	{
		relative_cell->z++;
		return true;
	}
	relative_cell->z = -1;
	
	return false;
}

/** next_cell2D
 * Function used for finding the next cell when using spatial partitioning. Z component is ignored
 * Upddates the relative cell variable which can have value of -1, 0 or +1
 * @param relative_cell pointer to the relative cell position
 * @return boolean if there is a next cell. True unless relative_Cell value was 1,1
 */
__device__ bool next_cell2D(glm::ivec3* relative_cell)
{
	if (relative_cell->x < 1)
	{
		relative_cell->x++;
		return true;
	}
	relative_cell->x = -1;

	if (relative_cell->y < 1)
	{
		relative_cell->y++;
		return true;
	}
	relative_cell->y = -1;
	
	return false;
}


/** Quiescent2Proliferating_function_filter
 *	Standard agent condition function. Filters agents from one state list to the next depending on the condition
 * @param currentState xmachine_memory_Fibroblast_list representing agent i the current state
 * @param nextState xmachine_memory_Fibroblast_list representing agent i the next state
 */
 __global__ void Quiescent2Proliferating_function_filter(xmachine_memory_Fibroblast_list* currentState, xmachine_memory_Fibroblast_list* nextState)
 {
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;
	
	//check thread max
	if (index < d_xmachine_memory_Fibroblast_count){
	
		//apply the filter
		if (currentState->proliferate_bool[index]==1)
		{	//copy agent data to newstate list
			nextState->id[index] = currentState->id[index];
			nextState->x[index] = currentState->x[index];
			nextState->y[index] = currentState->y[index];
			nextState->z[index] = currentState->z[index];
			nextState->doublings[index] = currentState->doublings[index];
			nextState->damage[index] = currentState->damage[index];
			nextState->proliferate_bool[index] = currentState->proliferate_bool[index];
			nextState->transition_to_early_sen[index] = currentState->transition_to_early_sen[index];
			//set scan input flag to 1
			nextState->_scan_input[index] = 1;
		}
		else
		{
			//set scan input flag of current state to 1 (keep agent)
			currentState->_scan_input[index] = 1;
		}
	
	}
 }

/** TransitionToEarlySen_function_filter
 *	Standard agent condition function. Filters agents from one state list to the next depending on the condition
 * @param currentState xmachine_memory_Fibroblast_list representing agent i the current state
 * @param nextState xmachine_memory_Fibroblast_list representing agent i the next state
 */
 __global__ void TransitionToEarlySen_function_filter(xmachine_memory_Fibroblast_list* currentState, xmachine_memory_Fibroblast_list* nextState)
 {
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;
	
	//check thread max
	if (index < d_xmachine_memory_Fibroblast_count){
	
		//apply the filter
		if (currentState->transition_to_early_sen[index]==1)
		{	//copy agent data to newstate list
			nextState->id[index] = currentState->id[index];
			nextState->x[index] = currentState->x[index];
			nextState->y[index] = currentState->y[index];
			nextState->z[index] = currentState->z[index];
			nextState->doublings[index] = currentState->doublings[index];
			nextState->damage[index] = currentState->damage[index];
			nextState->proliferate_bool[index] = currentState->proliferate_bool[index];
			nextState->transition_to_early_sen[index] = currentState->transition_to_early_sen[index];
			//set scan input flag to 1
			nextState->_scan_input[index] = 1;
		}
		else
		{
			//set scan input flag of current state to 1 (keep agent)
			currentState->_scan_input[index] = 1;
		}
	
	}
 }

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created TissueBlock agent functions */

/** reset_TissueBlock_scan_input
 * TissueBlock agent reset scan input function
 * @param agents The xmachine_memory_TissueBlock_list agent list
 */
__global__ void reset_TissueBlock_scan_input(xmachine_memory_TissueBlock_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_TissueBlock_Agents
 * TissueBlock scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_TissueBlock_list agent list destination
 * @param agents_src xmachine_memory_TissueBlock_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_TissueBlock_Agents(xmachine_memory_TissueBlock_list* agents_dst, xmachine_memory_TissueBlock_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->id[output_index] = agents_src->id[index];        
		agents_dst->x[output_index] = agents_src->x[index];        
		agents_dst->y[output_index] = agents_src->y[index];        
		agents_dst->z[output_index] = agents_src->z[index];        
		agents_dst->damage[output_index] = agents_src->damage[index];
	}
}

/** append_TissueBlock_Agents
 * TissueBlock scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_TissueBlock_list agent list destination
 * @param agents_src xmachine_memory_TissueBlock_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_TissueBlock_Agents(xmachine_memory_TissueBlock_list* agents_dst, xmachine_memory_TissueBlock_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->id[output_index] = agents_src->id[index];
	    agents_dst->x[output_index] = agents_src->x[index];
	    agents_dst->y[output_index] = agents_src->y[index];
	    agents_dst->z[output_index] = agents_src->z[index];
	    agents_dst->damage[output_index] = agents_src->damage[index];
    }
}

/** add_TissueBlock_agent
 * Continuous TissueBlock agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_TissueBlock_list to add agents to 
 * @param id agent variable of type int
 * @param x agent variable of type float
 * @param y agent variable of type float
 * @param z agent variable of type float
 * @param damage agent variable of type int
 */
template <int AGENT_TYPE>
__device__ void add_TissueBlock_agent(xmachine_memory_TissueBlock_list* agents, int id, float x, float y, float z, int damage){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->id[index] = id;
	agents->x[index] = x;
	agents->y[index] = y;
	agents->z[index] = z;
	agents->damage[index] = damage;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_TissueBlock_agent(xmachine_memory_TissueBlock_list* agents, int id, float x, float y, float z, int damage){
    add_TissueBlock_agent<DISCRETE_2D>(agents, id, x, y, z, damage);
}

/** reorder_TissueBlock_agents
 * Continuous TissueBlock agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_TissueBlock_agents(unsigned int* values, xmachine_memory_TissueBlock_list* unordered_agents, xmachine_memory_TissueBlock_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->id[index] = unordered_agents->id[old_pos];
	ordered_agents->x[index] = unordered_agents->x[old_pos];
	ordered_agents->y[index] = unordered_agents->y[old_pos];
	ordered_agents->z[index] = unordered_agents->z[old_pos];
	ordered_agents->damage[index] = unordered_agents->damage[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created Fibroblast agent functions */

/** reset_Fibroblast_scan_input
 * Fibroblast agent reset scan input function
 * @param agents The xmachine_memory_Fibroblast_list agent list
 */
__global__ void reset_Fibroblast_scan_input(xmachine_memory_Fibroblast_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_Fibroblast_Agents
 * Fibroblast scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Fibroblast_list agent list destination
 * @param agents_src xmachine_memory_Fibroblast_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_Fibroblast_Agents(xmachine_memory_Fibroblast_list* agents_dst, xmachine_memory_Fibroblast_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->id[output_index] = agents_src->id[index];        
		agents_dst->x[output_index] = agents_src->x[index];        
		agents_dst->y[output_index] = agents_src->y[index];        
		agents_dst->z[output_index] = agents_src->z[index];        
		agents_dst->doublings[output_index] = agents_src->doublings[index];        
		agents_dst->damage[output_index] = agents_src->damage[index];        
		agents_dst->proliferate_bool[output_index] = agents_src->proliferate_bool[index];        
		agents_dst->transition_to_early_sen[output_index] = agents_src->transition_to_early_sen[index];
	}
}

/** append_Fibroblast_Agents
 * Fibroblast scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Fibroblast_list agent list destination
 * @param agents_src xmachine_memory_Fibroblast_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_Fibroblast_Agents(xmachine_memory_Fibroblast_list* agents_dst, xmachine_memory_Fibroblast_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->id[output_index] = agents_src->id[index];
	    agents_dst->x[output_index] = agents_src->x[index];
	    agents_dst->y[output_index] = agents_src->y[index];
	    agents_dst->z[output_index] = agents_src->z[index];
	    agents_dst->doublings[output_index] = agents_src->doublings[index];
	    agents_dst->damage[output_index] = agents_src->damage[index];
	    agents_dst->proliferate_bool[output_index] = agents_src->proliferate_bool[index];
	    agents_dst->transition_to_early_sen[output_index] = agents_src->transition_to_early_sen[index];
    }
}

/** add_Fibroblast_agent
 * Continuous Fibroblast agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_Fibroblast_list to add agents to 
 * @param id agent variable of type int
 * @param x agent variable of type float
 * @param y agent variable of type float
 * @param z agent variable of type float
 * @param doublings agent variable of type float
 * @param damage agent variable of type int
 * @param proliferate_bool agent variable of type int
 * @param transition_to_early_sen agent variable of type int
 */
template <int AGENT_TYPE>
__device__ void add_Fibroblast_agent(xmachine_memory_Fibroblast_list* agents, int id, float x, float y, float z, float doublings, int damage, int proliferate_bool, int transition_to_early_sen){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->id[index] = id;
	agents->x[index] = x;
	agents->y[index] = y;
	agents->z[index] = z;
	agents->doublings[index] = doublings;
	agents->damage[index] = damage;
	agents->proliferate_bool[index] = proliferate_bool;
	agents->transition_to_early_sen[index] = transition_to_early_sen;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_Fibroblast_agent(xmachine_memory_Fibroblast_list* agents, int id, float x, float y, float z, float doublings, int damage, int proliferate_bool, int transition_to_early_sen){
    add_Fibroblast_agent<DISCRETE_2D>(agents, id, x, y, z, doublings, damage, proliferate_bool, transition_to_early_sen);
}

/** reorder_Fibroblast_agents
 * Continuous Fibroblast agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_Fibroblast_agents(unsigned int* values, xmachine_memory_Fibroblast_list* unordered_agents, xmachine_memory_Fibroblast_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->id[index] = unordered_agents->id[old_pos];
	ordered_agents->x[index] = unordered_agents->x[old_pos];
	ordered_agents->y[index] = unordered_agents->y[old_pos];
	ordered_agents->z[index] = unordered_agents->z[old_pos];
	ordered_agents->doublings[index] = unordered_agents->doublings[old_pos];
	ordered_agents->damage[index] = unordered_agents->damage[old_pos];
	ordered_agents->proliferate_bool[index] = unordered_agents->proliferate_bool[old_pos];
	ordered_agents->transition_to_early_sen[index] = unordered_agents->transition_to_early_sen[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created fibroblast_damage_report message functions */


/** add_fibroblast_damage_report_message
 * Add non partitioned or spatially partitioned fibroblast_damage_report message
 * @param messages xmachine_message_fibroblast_damage_report_list message list to add too
 * @param id agent variable of type int
 * @param x agent variable of type float
 * @param y agent variable of type float
 * @param z agent variable of type float
 * @param damage agent variable of type int
 */
__device__ void add_fibroblast_damage_report_message(xmachine_message_fibroblast_damage_report_list* messages, int id, float x, float y, float z, int damage){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_fibroblast_damage_report_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_fibroblast_damage_report_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_fibroblast_damage_report_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_fibroblast_damage_report Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->id[index] = id;
	messages->x[index] = x;
	messages->y[index] = y;
	messages->z[index] = z;
	messages->damage[index] = damage;

}

/**
 * Scatter non partitioned or spatially partitioned fibroblast_damage_report message (for optional messages)
 * @param messages scatter_optional_fibroblast_damage_report_messages Sparse xmachine_message_fibroblast_damage_report_list message list
 * @param message_swap temp xmachine_message_fibroblast_damage_report_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_fibroblast_damage_report_messages(xmachine_message_fibroblast_damage_report_list* messages, xmachine_message_fibroblast_damage_report_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_fibroblast_damage_report_count;

		//AoS - xmachine_message_fibroblast_damage_report Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->id[output_index] = messages_swap->id[index];
		messages->x[output_index] = messages_swap->x[index];
		messages->y[output_index] = messages_swap->y[index];
		messages->z[output_index] = messages_swap->z[index];
		messages->damage[output_index] = messages_swap->damage[index];				
	}
}

/** reset_fibroblast_damage_report_swaps
 * Reset non partitioned or spatially partitioned fibroblast_damage_report message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_fibroblast_damage_report_swaps(xmachine_message_fibroblast_damage_report_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

/** message_fibroblast_damage_report_grid_position
 * Calculates the grid cell position given an glm::vec3 vector
 * @param position glm::vec3 vector representing a position
 */
__device__ glm::ivec3 message_fibroblast_damage_report_grid_position(glm::vec3 position)
{
    glm::ivec3 gridPos;
    gridPos.x = floor((position.x - d_message_fibroblast_damage_report_min_bounds.x) * (float)d_message_fibroblast_damage_report_partitionDim.x / (d_message_fibroblast_damage_report_max_bounds.x - d_message_fibroblast_damage_report_min_bounds.x));
    gridPos.y = floor((position.y - d_message_fibroblast_damage_report_min_bounds.y) * (float)d_message_fibroblast_damage_report_partitionDim.y / (d_message_fibroblast_damage_report_max_bounds.y - d_message_fibroblast_damage_report_min_bounds.y));
    gridPos.z = floor((position.z - d_message_fibroblast_damage_report_min_bounds.z) * (float)d_message_fibroblast_damage_report_partitionDim.z / (d_message_fibroblast_damage_report_max_bounds.z - d_message_fibroblast_damage_report_min_bounds.z));

	//do wrapping or bounding
	

    return gridPos;
}

/** message_fibroblast_damage_report_hash
 * Given the grid position in partition space this function calculates a hash value
 * @param gridPos The position in partition space
 */
__device__ unsigned int message_fibroblast_damage_report_hash(glm::ivec3 gridPos)
{
	//cheap bounding without mod (within range +- partition dimension)
	gridPos.x = (gridPos.x<0)? d_message_fibroblast_damage_report_partitionDim.x-1: gridPos.x; 
	gridPos.x = (gridPos.x>=d_message_fibroblast_damage_report_partitionDim.x)? 0 : gridPos.x; 
	gridPos.y = (gridPos.y<0)? d_message_fibroblast_damage_report_partitionDim.y-1 : gridPos.y; 
	gridPos.y = (gridPos.y>=d_message_fibroblast_damage_report_partitionDim.y)? 0 : gridPos.y; 
	gridPos.z = (gridPos.z<0)? d_message_fibroblast_damage_report_partitionDim.z-1: gridPos.z; 
	gridPos.z = (gridPos.z>=d_message_fibroblast_damage_report_partitionDim.z)? 0 : gridPos.z; 

	//unique id
	return ((gridPos.z * d_message_fibroblast_damage_report_partitionDim.y) * d_message_fibroblast_damage_report_partitionDim.x) + (gridPos.y * d_message_fibroblast_damage_report_partitionDim.x) + gridPos.x;
}

#ifdef FAST_ATOMIC_SORTING
	/** hist_fibroblast_damage_report_messages
		 * Kernal function for performing a histogram (count) on each partition bin and saving the hash and index of a message within that bin
		 * @param local_bin_index output index of the message within the calculated bin
		 * @param unsorted_index output bin index (hash) value
		 * @param messages the message list used to generate the hash value outputs
		 * @param agent_count the current number of agents outputting messages
		 */
	__global__ void hist_fibroblast_damage_report_messages(uint* local_bin_index, uint* unsorted_index, int* global_bin_count, xmachine_message_fibroblast_damage_report_list* messages, int agent_count)
	{
		unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		if (index >= agent_count)
			return;
        glm::vec3 position = glm::vec3(messages->x[index], messages->y[index], messages->z[index]);
		glm::ivec3 grid_position = message_fibroblast_damage_report_grid_position(position);
		unsigned int hash = message_fibroblast_damage_report_hash(grid_position);
		unsigned int bin_idx = atomicInc((unsigned int*) &global_bin_count[hash], 0xFFFFFFFF);
		local_bin_index[index] = bin_idx;
		unsorted_index[index] = hash;
	}
	
	/** reorder_fibroblast_damage_report_messages
	 * Reorders the messages accoring to the partition boundary matrix start indices of each bin
	 * @param local_bin_index index of the message within the desired bin
	 * @param unsorted_index bin index (hash) value
	 * @param pbm_start_index the start indices of the partition boundary matrix
	 * @param unordered_messages the original unordered message data
	 * @param ordered_messages buffer used to scatter messages into the correct order
	  @param agent_count the current number of agents outputting messages
	 */
	 __global__ void reorder_fibroblast_damage_report_messages(uint* local_bin_index, uint* unsorted_index, int* pbm_start_index, xmachine_message_fibroblast_damage_report_list* unordered_messages, xmachine_message_fibroblast_damage_report_list* ordered_messages, int agent_count)
	{
		int index = (blockIdx.x *blockDim.x) + threadIdx.x;

		if (index >= agent_count)
			return;

		uint i = unsorted_index[index];
		unsigned int sorted_index = local_bin_index[index] + pbm_start_index[i];

		//finally reorder agent data
		ordered_messages->id[sorted_index] = unordered_messages->id[index];
		ordered_messages->x[sorted_index] = unordered_messages->x[index];
		ordered_messages->y[sorted_index] = unordered_messages->y[index];
		ordered_messages->z[sorted_index] = unordered_messages->z[index];
		ordered_messages->damage[sorted_index] = unordered_messages->damage[index];
	}
	 
#else

	/** hash_fibroblast_damage_report_messages
	 * Kernal function for calculating a hash value for each messahe depending on its position
	 * @param keys output for the hash key
	 * @param values output for the index value
	 * @param messages the message list used to generate the hash value outputs
	 */
	__global__ void hash_fibroblast_damage_report_messages(uint* keys, uint* values, xmachine_message_fibroblast_damage_report_list* messages)
	{
		unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
        glm::vec3 position = glm::vec3(messages->x[index], messages->y[index], messages->z[index]);
		glm::ivec3 grid_position = message_fibroblast_damage_report_grid_position(position);
		unsigned int hash = message_fibroblast_damage_report_hash(grid_position);

		keys[index] = hash;
		values[index] = index;
	}

	/** reorder_fibroblast_damage_report_messages
	 * Reorders the messages accoring to the ordered sort identifiers and builds a Partition Boundary Matrix by looking at the previosu threads sort id.
	 * @param keys the sorted hash keys
	 * @param values the sorted index values
	 * @param matrix the PBM
	 * @param unordered_messages the original unordered message data
	 * @param ordered_messages buffer used to scatter messages into the correct order
	 */
	__global__ void reorder_fibroblast_damage_report_messages(uint* keys, uint* values, xmachine_message_fibroblast_damage_report_PBM* matrix, xmachine_message_fibroblast_damage_report_list* unordered_messages, xmachine_message_fibroblast_damage_report_list* ordered_messages)
	{
		extern __shared__ int sm_data [];

		int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		//load threads sort key into sm
		uint key = keys[index];
		uint old_pos = values[index];

		sm_data[threadIdx.x] = key;
		__syncthreads();
	
		unsigned int prev_key;

		//if first thread then no prev sm value so get prev from global memory 
		if (threadIdx.x == 0)
		{
			//first thread has no prev value so ignore
			if (index != 0)
				prev_key = keys[index-1];
		}
		//get previous ident from sm
		else	
		{
			prev_key = sm_data[threadIdx.x-1];
		}

		//TODO: Check key is not out of bounds

		//set partition boundaries
		if (index < d_message_fibroblast_damage_report_count)
		{
			//if first thread then set first partition cell start
			if (index == 0)
			{
				matrix->start[key] = index;
			}

			//if edge of a boundr update start and end of partition
			else if (prev_key != key)
			{
				//set start for key
				matrix->start[key] = index;

				//set end for key -1
				matrix->end_or_count[prev_key] = index;
			}

			//if last thread then set final partition cell end
			if (index == d_message_fibroblast_damage_report_count-1)
			{
				matrix->end_or_count[key] = index+1;
			}
		}
	
		//finally reorder agent data
		ordered_messages->id[index] = unordered_messages->id[old_pos];
		ordered_messages->x[index] = unordered_messages->x[old_pos];
		ordered_messages->y[index] = unordered_messages->y[old_pos];
		ordered_messages->z[index] = unordered_messages->z[old_pos];
		ordered_messages->damage[index] = unordered_messages->damage[old_pos];
	}

#endif

/** load_next_fibroblast_damage_report_message
 * Used to load the next message data to shared memory
 * Idea is check the current cell index to see if we can simply get a message from the current cell
 * If we are at the end of the current cell then loop till we find the next cell with messages (this way we ignore cells with no messages)
 * @param messages the message list
 * @param partition_matrix the PBM
 * @param relative_cell the relative partition cell position from the agent position
 * @param cell_index_max the maximum index of the current partition cell
 * @param agent_grid_cell the agents partition cell position
 * @param cell_index the current cell index in agent_grid_cell+relative_cell
 * @return true if a message has been loaded into sm false otherwise
 */
__device__ bool load_next_fibroblast_damage_report_message(xmachine_message_fibroblast_damage_report_list* messages, xmachine_message_fibroblast_damage_report_PBM* partition_matrix, glm::ivec3 relative_cell, int cell_index_max, glm::ivec3 agent_grid_cell, int cell_index)
{
	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];

	int move_cell = true;
	cell_index ++;

	//see if we need to move to a new partition cell
	if(cell_index < cell_index_max)
		move_cell = false;

	while(move_cell)
	{
		//get the next relative grid position 
        if (next_cell3D(&relative_cell))
		{
			//calculate the next cells grid position and hash
			glm::ivec3 next_cell_position = agent_grid_cell + relative_cell;
			int next_cell_hash = message_fibroblast_damage_report_hash(next_cell_position);
			//use the hash to calculate the start index
			int cell_index_min = tex1Dfetch(tex_xmachine_message_fibroblast_damage_report_pbm_start, next_cell_hash + d_tex_xmachine_message_fibroblast_damage_report_pbm_start_offset);
			cell_index_max = tex1Dfetch(tex_xmachine_message_fibroblast_damage_report_pbm_end_or_count, next_cell_hash + d_tex_xmachine_message_fibroblast_damage_report_pbm_end_or_count_offset);
			//check for messages in the cell (cell index max is the count for atomic sorting)
#ifdef FAST_ATOMIC_SORTING
			if (cell_index_max > 0)
			{
				//when using fast atomics value represents bin count not last index!
				cell_index_max += cell_index_min; //when using fast atomics value represents bin count not last index!
#else
			if (cell_index_min != 0xffffffff)
			{
#endif
				//start from the cell index min
				cell_index = cell_index_min;
				//exit the loop as we have found a valid cell with message data
				move_cell = false;
			}
		}
		else
		{
			//we have exhausted all the neighbouring cells so there are no more messages
			return false;
		}
	}
	
	//get message data using texture fetch
	xmachine_message_fibroblast_damage_report temp_message;
	temp_message._relative_cell = relative_cell;
	temp_message._cell_index_max = cell_index_max;
	temp_message._cell_index = cell_index;
	temp_message._agent_grid_cell = agent_grid_cell;

	//Using texture cache
  temp_message.id = tex1Dfetch(tex_xmachine_message_fibroblast_damage_report_id, cell_index + d_tex_xmachine_message_fibroblast_damage_report_id_offset); temp_message.x = tex1Dfetch(tex_xmachine_message_fibroblast_damage_report_x, cell_index + d_tex_xmachine_message_fibroblast_damage_report_x_offset); temp_message.y = tex1Dfetch(tex_xmachine_message_fibroblast_damage_report_y, cell_index + d_tex_xmachine_message_fibroblast_damage_report_y_offset); temp_message.z = tex1Dfetch(tex_xmachine_message_fibroblast_damage_report_z, cell_index + d_tex_xmachine_message_fibroblast_damage_report_z_offset); temp_message.damage = tex1Dfetch(tex_xmachine_message_fibroblast_damage_report_damage, cell_index + d_tex_xmachine_message_fibroblast_damage_report_damage_offset); 

	//load it into shared memory (no sync as no sharing between threads)
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_fibroblast_damage_report));
	xmachine_message_fibroblast_damage_report* sm_message = ((xmachine_message_fibroblast_damage_report*)&message_share[message_index]);
	sm_message[0] = temp_message;

	return true;
}


/*
 * get first spatial partitioned fibroblast_damage_report message (first batch load into shared memory)
 */
__device__ xmachine_message_fibroblast_damage_report* get_first_fibroblast_damage_report_message(xmachine_message_fibroblast_damage_report_list* messages, xmachine_message_fibroblast_damage_report_PBM* partition_matrix, float x, float y, float z){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];

	// If there are no messages, do not load any messages
	if(d_message_fibroblast_damage_report_count == 0){
		return nullptr;
	}

	glm::ivec3 relative_cell = glm::ivec3(-2, -1, -1);
	int cell_index_max = 0;
	int cell_index = 0;
	glm::vec3 position = glm::vec3(x, y, z);
	glm::ivec3 agent_grid_cell = message_fibroblast_damage_report_grid_position(position);
	
	if (load_next_fibroblast_damage_report_message(messages, partition_matrix, relative_cell, cell_index_max, agent_grid_cell, cell_index))
	{
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_fibroblast_damage_report));
		return ((xmachine_message_fibroblast_damage_report*)&message_share[message_index]);
	}
	else
	{
		return nullptr;
	}
}

/*
 * get next spatial partitioned fibroblast_damage_report message (either from SM or next batch load)
 */
__device__ xmachine_message_fibroblast_damage_report* get_next_fibroblast_damage_report_message(xmachine_message_fibroblast_damage_report* message, xmachine_message_fibroblast_damage_report_list* messages, xmachine_message_fibroblast_damage_report_PBM* partition_matrix){
	
	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	// If there are no messages, do not load any messages
	if(d_message_fibroblast_damage_report_count == 0){
		return nullptr;
	}
	
	if (load_next_fibroblast_damage_report_message(messages, partition_matrix, message->_relative_cell, message->_cell_index_max, message->_agent_grid_cell, message->_cell_index))
	{
		//get conflict free address of 
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_fibroblast_damage_report));
		return ((xmachine_message_fibroblast_damage_report*)&message_share[message_index]);
	}
	else
		return nullptr;
	
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created tissue_damage_report message functions */


/** add_tissue_damage_report_message
 * Add non partitioned or spatially partitioned tissue_damage_report message
 * @param messages xmachine_message_tissue_damage_report_list message list to add too
 * @param id agent variable of type int
 * @param x agent variable of type float
 * @param y agent variable of type float
 * @param z agent variable of type float
 * @param damage agent variable of type int
 */
__device__ void add_tissue_damage_report_message(xmachine_message_tissue_damage_report_list* messages, int id, float x, float y, float z, int damage){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_tissue_damage_report_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_tissue_damage_report_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_tissue_damage_report_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_tissue_damage_report Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->id[index] = id;
	messages->x[index] = x;
	messages->y[index] = y;
	messages->z[index] = z;
	messages->damage[index] = damage;

}

/**
 * Scatter non partitioned or spatially partitioned tissue_damage_report message (for optional messages)
 * @param messages scatter_optional_tissue_damage_report_messages Sparse xmachine_message_tissue_damage_report_list message list
 * @param message_swap temp xmachine_message_tissue_damage_report_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_tissue_damage_report_messages(xmachine_message_tissue_damage_report_list* messages, xmachine_message_tissue_damage_report_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_tissue_damage_report_count;

		//AoS - xmachine_message_tissue_damage_report Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->id[output_index] = messages_swap->id[index];
		messages->x[output_index] = messages_swap->x[index];
		messages->y[output_index] = messages_swap->y[index];
		messages->z[output_index] = messages_swap->z[index];
		messages->damage[output_index] = messages_swap->damage[index];				
	}
}

/** reset_tissue_damage_report_swaps
 * Reset non partitioned or spatially partitioned tissue_damage_report message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_tissue_damage_report_swaps(xmachine_message_tissue_damage_report_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

/** message_tissue_damage_report_grid_position
 * Calculates the grid cell position given an glm::vec3 vector
 * @param position glm::vec3 vector representing a position
 */
__device__ glm::ivec3 message_tissue_damage_report_grid_position(glm::vec3 position)
{
    glm::ivec3 gridPos;
    gridPos.x = floor((position.x - d_message_tissue_damage_report_min_bounds.x) * (float)d_message_tissue_damage_report_partitionDim.x / (d_message_tissue_damage_report_max_bounds.x - d_message_tissue_damage_report_min_bounds.x));
    gridPos.y = floor((position.y - d_message_tissue_damage_report_min_bounds.y) * (float)d_message_tissue_damage_report_partitionDim.y / (d_message_tissue_damage_report_max_bounds.y - d_message_tissue_damage_report_min_bounds.y));
    gridPos.z = floor((position.z - d_message_tissue_damage_report_min_bounds.z) * (float)d_message_tissue_damage_report_partitionDim.z / (d_message_tissue_damage_report_max_bounds.z - d_message_tissue_damage_report_min_bounds.z));

	//do wrapping or bounding
	

    return gridPos;
}

/** message_tissue_damage_report_hash
 * Given the grid position in partition space this function calculates a hash value
 * @param gridPos The position in partition space
 */
__device__ unsigned int message_tissue_damage_report_hash(glm::ivec3 gridPos)
{
	//cheap bounding without mod (within range +- partition dimension)
	gridPos.x = (gridPos.x<0)? d_message_tissue_damage_report_partitionDim.x-1: gridPos.x; 
	gridPos.x = (gridPos.x>=d_message_tissue_damage_report_partitionDim.x)? 0 : gridPos.x; 
	gridPos.y = (gridPos.y<0)? d_message_tissue_damage_report_partitionDim.y-1 : gridPos.y; 
	gridPos.y = (gridPos.y>=d_message_tissue_damage_report_partitionDim.y)? 0 : gridPos.y; 
	gridPos.z = (gridPos.z<0)? d_message_tissue_damage_report_partitionDim.z-1: gridPos.z; 
	gridPos.z = (gridPos.z>=d_message_tissue_damage_report_partitionDim.z)? 0 : gridPos.z; 

	//unique id
	return ((gridPos.z * d_message_tissue_damage_report_partitionDim.y) * d_message_tissue_damage_report_partitionDim.x) + (gridPos.y * d_message_tissue_damage_report_partitionDim.x) + gridPos.x;
}

#ifdef FAST_ATOMIC_SORTING
	/** hist_tissue_damage_report_messages
		 * Kernal function for performing a histogram (count) on each partition bin and saving the hash and index of a message within that bin
		 * @param local_bin_index output index of the message within the calculated bin
		 * @param unsorted_index output bin index (hash) value
		 * @param messages the message list used to generate the hash value outputs
		 * @param agent_count the current number of agents outputting messages
		 */
	__global__ void hist_tissue_damage_report_messages(uint* local_bin_index, uint* unsorted_index, int* global_bin_count, xmachine_message_tissue_damage_report_list* messages, int agent_count)
	{
		unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		if (index >= agent_count)
			return;
        glm::vec3 position = glm::vec3(messages->x[index], messages->y[index], messages->z[index]);
		glm::ivec3 grid_position = message_tissue_damage_report_grid_position(position);
		unsigned int hash = message_tissue_damage_report_hash(grid_position);
		unsigned int bin_idx = atomicInc((unsigned int*) &global_bin_count[hash], 0xFFFFFFFF);
		local_bin_index[index] = bin_idx;
		unsorted_index[index] = hash;
	}
	
	/** reorder_tissue_damage_report_messages
	 * Reorders the messages accoring to the partition boundary matrix start indices of each bin
	 * @param local_bin_index index of the message within the desired bin
	 * @param unsorted_index bin index (hash) value
	 * @param pbm_start_index the start indices of the partition boundary matrix
	 * @param unordered_messages the original unordered message data
	 * @param ordered_messages buffer used to scatter messages into the correct order
	  @param agent_count the current number of agents outputting messages
	 */
	 __global__ void reorder_tissue_damage_report_messages(uint* local_bin_index, uint* unsorted_index, int* pbm_start_index, xmachine_message_tissue_damage_report_list* unordered_messages, xmachine_message_tissue_damage_report_list* ordered_messages, int agent_count)
	{
		int index = (blockIdx.x *blockDim.x) + threadIdx.x;

		if (index >= agent_count)
			return;

		uint i = unsorted_index[index];
		unsigned int sorted_index = local_bin_index[index] + pbm_start_index[i];

		//finally reorder agent data
		ordered_messages->id[sorted_index] = unordered_messages->id[index];
		ordered_messages->x[sorted_index] = unordered_messages->x[index];
		ordered_messages->y[sorted_index] = unordered_messages->y[index];
		ordered_messages->z[sorted_index] = unordered_messages->z[index];
		ordered_messages->damage[sorted_index] = unordered_messages->damage[index];
	}
	 
#else

	/** hash_tissue_damage_report_messages
	 * Kernal function for calculating a hash value for each messahe depending on its position
	 * @param keys output for the hash key
	 * @param values output for the index value
	 * @param messages the message list used to generate the hash value outputs
	 */
	__global__ void hash_tissue_damage_report_messages(uint* keys, uint* values, xmachine_message_tissue_damage_report_list* messages)
	{
		unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
        glm::vec3 position = glm::vec3(messages->x[index], messages->y[index], messages->z[index]);
		glm::ivec3 grid_position = message_tissue_damage_report_grid_position(position);
		unsigned int hash = message_tissue_damage_report_hash(grid_position);

		keys[index] = hash;
		values[index] = index;
	}

	/** reorder_tissue_damage_report_messages
	 * Reorders the messages accoring to the ordered sort identifiers and builds a Partition Boundary Matrix by looking at the previosu threads sort id.
	 * @param keys the sorted hash keys
	 * @param values the sorted index values
	 * @param matrix the PBM
	 * @param unordered_messages the original unordered message data
	 * @param ordered_messages buffer used to scatter messages into the correct order
	 */
	__global__ void reorder_tissue_damage_report_messages(uint* keys, uint* values, xmachine_message_tissue_damage_report_PBM* matrix, xmachine_message_tissue_damage_report_list* unordered_messages, xmachine_message_tissue_damage_report_list* ordered_messages)
	{
		extern __shared__ int sm_data [];

		int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		//load threads sort key into sm
		uint key = keys[index];
		uint old_pos = values[index];

		sm_data[threadIdx.x] = key;
		__syncthreads();
	
		unsigned int prev_key;

		//if first thread then no prev sm value so get prev from global memory 
		if (threadIdx.x == 0)
		{
			//first thread has no prev value so ignore
			if (index != 0)
				prev_key = keys[index-1];
		}
		//get previous ident from sm
		else	
		{
			prev_key = sm_data[threadIdx.x-1];
		}

		//TODO: Check key is not out of bounds

		//set partition boundaries
		if (index < d_message_tissue_damage_report_count)
		{
			//if first thread then set first partition cell start
			if (index == 0)
			{
				matrix->start[key] = index;
			}

			//if edge of a boundr update start and end of partition
			else if (prev_key != key)
			{
				//set start for key
				matrix->start[key] = index;

				//set end for key -1
				matrix->end_or_count[prev_key] = index;
			}

			//if last thread then set final partition cell end
			if (index == d_message_tissue_damage_report_count-1)
			{
				matrix->end_or_count[key] = index+1;
			}
		}
	
		//finally reorder agent data
		ordered_messages->id[index] = unordered_messages->id[old_pos];
		ordered_messages->x[index] = unordered_messages->x[old_pos];
		ordered_messages->y[index] = unordered_messages->y[old_pos];
		ordered_messages->z[index] = unordered_messages->z[old_pos];
		ordered_messages->damage[index] = unordered_messages->damage[old_pos];
	}

#endif

/** load_next_tissue_damage_report_message
 * Used to load the next message data to shared memory
 * Idea is check the current cell index to see if we can simply get a message from the current cell
 * If we are at the end of the current cell then loop till we find the next cell with messages (this way we ignore cells with no messages)
 * @param messages the message list
 * @param partition_matrix the PBM
 * @param relative_cell the relative partition cell position from the agent position
 * @param cell_index_max the maximum index of the current partition cell
 * @param agent_grid_cell the agents partition cell position
 * @param cell_index the current cell index in agent_grid_cell+relative_cell
 * @return true if a message has been loaded into sm false otherwise
 */
__device__ bool load_next_tissue_damage_report_message(xmachine_message_tissue_damage_report_list* messages, xmachine_message_tissue_damage_report_PBM* partition_matrix, glm::ivec3 relative_cell, int cell_index_max, glm::ivec3 agent_grid_cell, int cell_index)
{
	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];

	int move_cell = true;
	cell_index ++;

	//see if we need to move to a new partition cell
	if(cell_index < cell_index_max)
		move_cell = false;

	while(move_cell)
	{
		//get the next relative grid position 
        if (next_cell3D(&relative_cell))
		{
			//calculate the next cells grid position and hash
			glm::ivec3 next_cell_position = agent_grid_cell + relative_cell;
			int next_cell_hash = message_tissue_damage_report_hash(next_cell_position);
			//use the hash to calculate the start index
			int cell_index_min = tex1Dfetch(tex_xmachine_message_tissue_damage_report_pbm_start, next_cell_hash + d_tex_xmachine_message_tissue_damage_report_pbm_start_offset);
			cell_index_max = tex1Dfetch(tex_xmachine_message_tissue_damage_report_pbm_end_or_count, next_cell_hash + d_tex_xmachine_message_tissue_damage_report_pbm_end_or_count_offset);
			//check for messages in the cell (cell index max is the count for atomic sorting)
#ifdef FAST_ATOMIC_SORTING
			if (cell_index_max > 0)
			{
				//when using fast atomics value represents bin count not last index!
				cell_index_max += cell_index_min; //when using fast atomics value represents bin count not last index!
#else
			if (cell_index_min != 0xffffffff)
			{
#endif
				//start from the cell index min
				cell_index = cell_index_min;
				//exit the loop as we have found a valid cell with message data
				move_cell = false;
			}
		}
		else
		{
			//we have exhausted all the neighbouring cells so there are no more messages
			return false;
		}
	}
	
	//get message data using texture fetch
	xmachine_message_tissue_damage_report temp_message;
	temp_message._relative_cell = relative_cell;
	temp_message._cell_index_max = cell_index_max;
	temp_message._cell_index = cell_index;
	temp_message._agent_grid_cell = agent_grid_cell;

	//Using texture cache
  temp_message.id = tex1Dfetch(tex_xmachine_message_tissue_damage_report_id, cell_index + d_tex_xmachine_message_tissue_damage_report_id_offset); temp_message.x = tex1Dfetch(tex_xmachine_message_tissue_damage_report_x, cell_index + d_tex_xmachine_message_tissue_damage_report_x_offset); temp_message.y = tex1Dfetch(tex_xmachine_message_tissue_damage_report_y, cell_index + d_tex_xmachine_message_tissue_damage_report_y_offset); temp_message.z = tex1Dfetch(tex_xmachine_message_tissue_damage_report_z, cell_index + d_tex_xmachine_message_tissue_damage_report_z_offset); temp_message.damage = tex1Dfetch(tex_xmachine_message_tissue_damage_report_damage, cell_index + d_tex_xmachine_message_tissue_damage_report_damage_offset); 

	//load it into shared memory (no sync as no sharing between threads)
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_tissue_damage_report));
	xmachine_message_tissue_damage_report* sm_message = ((xmachine_message_tissue_damage_report*)&message_share[message_index]);
	sm_message[0] = temp_message;

	return true;
}


/*
 * get first spatial partitioned tissue_damage_report message (first batch load into shared memory)
 */
__device__ xmachine_message_tissue_damage_report* get_first_tissue_damage_report_message(xmachine_message_tissue_damage_report_list* messages, xmachine_message_tissue_damage_report_PBM* partition_matrix, float x, float y, float z){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];

	// If there are no messages, do not load any messages
	if(d_message_tissue_damage_report_count == 0){
		return nullptr;
	}

	glm::ivec3 relative_cell = glm::ivec3(-2, -1, -1);
	int cell_index_max = 0;
	int cell_index = 0;
	glm::vec3 position = glm::vec3(x, y, z);
	glm::ivec3 agent_grid_cell = message_tissue_damage_report_grid_position(position);
	
	if (load_next_tissue_damage_report_message(messages, partition_matrix, relative_cell, cell_index_max, agent_grid_cell, cell_index))
	{
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_tissue_damage_report));
		return ((xmachine_message_tissue_damage_report*)&message_share[message_index]);
	}
	else
	{
		return nullptr;
	}
}

/*
 * get next spatial partitioned tissue_damage_report message (either from SM or next batch load)
 */
__device__ xmachine_message_tissue_damage_report* get_next_tissue_damage_report_message(xmachine_message_tissue_damage_report* message, xmachine_message_tissue_damage_report_list* messages, xmachine_message_tissue_damage_report_PBM* partition_matrix){
	
	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	// If there are no messages, do not load any messages
	if(d_message_tissue_damage_report_count == 0){
		return nullptr;
	}
	
	if (load_next_tissue_damage_report_message(messages, partition_matrix, message->_relative_cell, message->_cell_index_max, message->_agent_grid_cell, message->_cell_index))
	{
		//get conflict free address of 
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_tissue_damage_report));
		return ((xmachine_message_tissue_damage_report*)&message_share[message_index]);
	}
	else
		return nullptr;
	
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created doublings message functions */


/** add_doublings_message
 * Add non partitioned or spatially partitioned doublings message
 * @param messages xmachine_message_doublings_list message list to add too
 * @param number agent variable of type int
 */
__device__ void add_doublings_message(xmachine_message_doublings_list* messages, int number){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_doublings_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_doublings_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_doublings_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_doublings Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->number[index] = number;

}

/**
 * Scatter non partitioned or spatially partitioned doublings message (for optional messages)
 * @param messages scatter_optional_doublings_messages Sparse xmachine_message_doublings_list message list
 * @param message_swap temp xmachine_message_doublings_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_doublings_messages(xmachine_message_doublings_list* messages, xmachine_message_doublings_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_doublings_count;

		//AoS - xmachine_message_doublings Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->number[output_index] = messages_swap->number[index];				
	}
}

/** reset_doublings_swaps
 * Reset non partitioned or spatially partitioned doublings message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_doublings_swaps(xmachine_message_doublings_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

__device__ xmachine_message_doublings* get_first_doublings_message(xmachine_message_doublings_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = (ceil((float)d_message_doublings_count/ blockDim.x)* blockDim.x);

	//if no messages then return a null pointer (false)
	if (wrap_size == 0)
		return nullptr;

	//global thread index
	int global_index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//global thread index
	int index = WRAP(global_index, wrap_size);

	//SoA to AoS - xmachine_message_doublings Coalesced memory read
	xmachine_message_doublings temp_message;
	temp_message._position = messages->_position[index];
	temp_message.number = messages->number[index];

	//AoS to shared memory
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_doublings));
	xmachine_message_doublings* sm_message = ((xmachine_message_doublings*)&message_share[message_index]);
	sm_message[0] = temp_message;

	__syncthreads();

  //HACK FOR 64 bit addressing issue in sm
	return ((xmachine_message_doublings*)&message_share[d_SM_START]);
}

__device__ xmachine_message_doublings* get_next_doublings_message(xmachine_message_doublings* message, xmachine_message_doublings_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = ceil((float)d_message_doublings_count/ blockDim.x)*blockDim.x;

	int i = WRAP((message->_position + 1),wrap_size);

	//If end of messages (last message not multiple of gridsize) go to 0 index
	if (i >= d_message_doublings_count)
		i = 0;

	//Check if back to start position of first message
	if (i == WRAP((blockDim.x* blockIdx.x), wrap_size))
		return nullptr;

	int tile = floor((float)i/(blockDim.x)); //tile is round down position over blockDim
	i = i % blockDim.x;						 //mod i for shared memory index

	//if count == Block Size load next tile int shared memory values
	if (i == 0){
		__syncthreads();					//make sure we don't change shared memory until all threads are here (important for emu-debug mode)
		
		//SoA to AoS - xmachine_message_doublings Coalesced memory read
		int index = (tile* blockDim.x) + threadIdx.x;
		xmachine_message_doublings temp_message;
		temp_message._position = messages->_position[index];
		temp_message.number = messages->number[index];

		//AoS to shared memory
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_doublings));
		xmachine_message_doublings* sm_message = ((xmachine_message_doublings*)&message_share[message_index]);
		sm_message[0] = temp_message;

		__syncthreads();					//make sure we don't start returning messages until all threads have updated shared memory
	}

	int message_index = SHARE_INDEX(i, sizeof(xmachine_message_doublings));
	return ((xmachine_message_doublings*)&message_share[message_index]);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created count message functions */


/** add_count_message
 * Add non partitioned or spatially partitioned count message
 * @param messages xmachine_message_count_list message list to add too
 * @param number agent variable of type int
 */
__device__ void add_count_message(xmachine_message_count_list* messages, int number){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_count_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_count_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_count_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_count Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->number[index] = number;

}

/**
 * Scatter non partitioned or spatially partitioned count message (for optional messages)
 * @param messages scatter_optional_count_messages Sparse xmachine_message_count_list message list
 * @param message_swap temp xmachine_message_count_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_count_messages(xmachine_message_count_list* messages, xmachine_message_count_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_count_count;

		//AoS - xmachine_message_count Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->number[output_index] = messages_swap->number[index];				
	}
}

/** reset_count_swaps
 * Reset non partitioned or spatially partitioned count message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_count_swaps(xmachine_message_count_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

__device__ xmachine_message_count* get_first_count_message(xmachine_message_count_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = (ceil((float)d_message_count_count/ blockDim.x)* blockDim.x);

	//if no messages then return a null pointer (false)
	if (wrap_size == 0)
		return nullptr;

	//global thread index
	int global_index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//global thread index
	int index = WRAP(global_index, wrap_size);

	//SoA to AoS - xmachine_message_count Coalesced memory read
	xmachine_message_count temp_message;
	temp_message._position = messages->_position[index];
	temp_message.number = messages->number[index];

	//AoS to shared memory
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_count));
	xmachine_message_count* sm_message = ((xmachine_message_count*)&message_share[message_index]);
	sm_message[0] = temp_message;

	__syncthreads();

  //HACK FOR 64 bit addressing issue in sm
	return ((xmachine_message_count*)&message_share[d_SM_START]);
}

__device__ xmachine_message_count* get_next_count_message(xmachine_message_count* message, xmachine_message_count_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = ceil((float)d_message_count_count/ blockDim.x)*blockDim.x;

	int i = WRAP((message->_position + 1),wrap_size);

	//If end of messages (last message not multiple of gridsize) go to 0 index
	if (i >= d_message_count_count)
		i = 0;

	//Check if back to start position of first message
	if (i == WRAP((blockDim.x* blockIdx.x), wrap_size))
		return nullptr;

	int tile = floor((float)i/(blockDim.x)); //tile is round down position over blockDim
	i = i % blockDim.x;						 //mod i for shared memory index

	//if count == Block Size load next tile int shared memory values
	if (i == 0){
		__syncthreads();					//make sure we don't change shared memory until all threads are here (important for emu-debug mode)
		
		//SoA to AoS - xmachine_message_count Coalesced memory read
		int index = (tile* blockDim.x) + threadIdx.x;
		xmachine_message_count temp_message;
		temp_message._position = messages->_position[index];
		temp_message.number = messages->number[index];

		//AoS to shared memory
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_count));
		xmachine_message_count* sm_message = ((xmachine_message_count*)&message_share[message_index]);
		sm_message[0] = temp_message;

		__syncthreads();					//make sure we don't start returning messages until all threads have updated shared memory
	}

	int message_index = SHARE_INDEX(i, sizeof(xmachine_message_count));
	return ((xmachine_message_count*)&message_share[message_index]);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created quiescent_location_report message functions */


/** add_quiescent_location_report_message
 * Add non partitioned or spatially partitioned quiescent_location_report message
 * @param messages xmachine_message_quiescent_location_report_list message list to add too
 * @param id agent variable of type int
 * @param x agent variable of type float
 * @param y agent variable of type float
 * @param z agent variable of type float
 */
__device__ void add_quiescent_location_report_message(xmachine_message_quiescent_location_report_list* messages, int id, float x, float y, float z){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_quiescent_location_report_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_quiescent_location_report_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_quiescent_location_report_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_quiescent_location_report Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->id[index] = id;
	messages->x[index] = x;
	messages->y[index] = y;
	messages->z[index] = z;

}

/**
 * Scatter non partitioned or spatially partitioned quiescent_location_report message (for optional messages)
 * @param messages scatter_optional_quiescent_location_report_messages Sparse xmachine_message_quiescent_location_report_list message list
 * @param message_swap temp xmachine_message_quiescent_location_report_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_quiescent_location_report_messages(xmachine_message_quiescent_location_report_list* messages, xmachine_message_quiescent_location_report_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_quiescent_location_report_count;

		//AoS - xmachine_message_quiescent_location_report Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->id[output_index] = messages_swap->id[index];
		messages->x[output_index] = messages_swap->x[index];
		messages->y[output_index] = messages_swap->y[index];
		messages->z[output_index] = messages_swap->z[index];				
	}
}

/** reset_quiescent_location_report_swaps
 * Reset non partitioned or spatially partitioned quiescent_location_report message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_quiescent_location_report_swaps(xmachine_message_quiescent_location_report_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

/** message_quiescent_location_report_grid_position
 * Calculates the grid cell position given an glm::vec3 vector
 * @param position glm::vec3 vector representing a position
 */
__device__ glm::ivec3 message_quiescent_location_report_grid_position(glm::vec3 position)
{
    glm::ivec3 gridPos;
    gridPos.x = floor((position.x - d_message_quiescent_location_report_min_bounds.x) * (float)d_message_quiescent_location_report_partitionDim.x / (d_message_quiescent_location_report_max_bounds.x - d_message_quiescent_location_report_min_bounds.x));
    gridPos.y = floor((position.y - d_message_quiescent_location_report_min_bounds.y) * (float)d_message_quiescent_location_report_partitionDim.y / (d_message_quiescent_location_report_max_bounds.y - d_message_quiescent_location_report_min_bounds.y));
    gridPos.z = floor((position.z - d_message_quiescent_location_report_min_bounds.z) * (float)d_message_quiescent_location_report_partitionDim.z / (d_message_quiescent_location_report_max_bounds.z - d_message_quiescent_location_report_min_bounds.z));

	//do wrapping or bounding
	

    return gridPos;
}

/** message_quiescent_location_report_hash
 * Given the grid position in partition space this function calculates a hash value
 * @param gridPos The position in partition space
 */
__device__ unsigned int message_quiescent_location_report_hash(glm::ivec3 gridPos)
{
	//cheap bounding without mod (within range +- partition dimension)
	gridPos.x = (gridPos.x<0)? d_message_quiescent_location_report_partitionDim.x-1: gridPos.x; 
	gridPos.x = (gridPos.x>=d_message_quiescent_location_report_partitionDim.x)? 0 : gridPos.x; 
	gridPos.y = (gridPos.y<0)? d_message_quiescent_location_report_partitionDim.y-1 : gridPos.y; 
	gridPos.y = (gridPos.y>=d_message_quiescent_location_report_partitionDim.y)? 0 : gridPos.y; 
	gridPos.z = (gridPos.z<0)? d_message_quiescent_location_report_partitionDim.z-1: gridPos.z; 
	gridPos.z = (gridPos.z>=d_message_quiescent_location_report_partitionDim.z)? 0 : gridPos.z; 

	//unique id
	return ((gridPos.z * d_message_quiescent_location_report_partitionDim.y) * d_message_quiescent_location_report_partitionDim.x) + (gridPos.y * d_message_quiescent_location_report_partitionDim.x) + gridPos.x;
}

#ifdef FAST_ATOMIC_SORTING
	/** hist_quiescent_location_report_messages
		 * Kernal function for performing a histogram (count) on each partition bin and saving the hash and index of a message within that bin
		 * @param local_bin_index output index of the message within the calculated bin
		 * @param unsorted_index output bin index (hash) value
		 * @param messages the message list used to generate the hash value outputs
		 * @param agent_count the current number of agents outputting messages
		 */
	__global__ void hist_quiescent_location_report_messages(uint* local_bin_index, uint* unsorted_index, int* global_bin_count, xmachine_message_quiescent_location_report_list* messages, int agent_count)
	{
		unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		if (index >= agent_count)
			return;
        glm::vec3 position = glm::vec3(messages->x[index], messages->y[index], messages->z[index]);
		glm::ivec3 grid_position = message_quiescent_location_report_grid_position(position);
		unsigned int hash = message_quiescent_location_report_hash(grid_position);
		unsigned int bin_idx = atomicInc((unsigned int*) &global_bin_count[hash], 0xFFFFFFFF);
		local_bin_index[index] = bin_idx;
		unsorted_index[index] = hash;
	}
	
	/** reorder_quiescent_location_report_messages
	 * Reorders the messages accoring to the partition boundary matrix start indices of each bin
	 * @param local_bin_index index of the message within the desired bin
	 * @param unsorted_index bin index (hash) value
	 * @param pbm_start_index the start indices of the partition boundary matrix
	 * @param unordered_messages the original unordered message data
	 * @param ordered_messages buffer used to scatter messages into the correct order
	  @param agent_count the current number of agents outputting messages
	 */
	 __global__ void reorder_quiescent_location_report_messages(uint* local_bin_index, uint* unsorted_index, int* pbm_start_index, xmachine_message_quiescent_location_report_list* unordered_messages, xmachine_message_quiescent_location_report_list* ordered_messages, int agent_count)
	{
		int index = (blockIdx.x *blockDim.x) + threadIdx.x;

		if (index >= agent_count)
			return;

		uint i = unsorted_index[index];
		unsigned int sorted_index = local_bin_index[index] + pbm_start_index[i];

		//finally reorder agent data
		ordered_messages->id[sorted_index] = unordered_messages->id[index];
		ordered_messages->x[sorted_index] = unordered_messages->x[index];
		ordered_messages->y[sorted_index] = unordered_messages->y[index];
		ordered_messages->z[sorted_index] = unordered_messages->z[index];
	}
	 
#else

	/** hash_quiescent_location_report_messages
	 * Kernal function for calculating a hash value for each messahe depending on its position
	 * @param keys output for the hash key
	 * @param values output for the index value
	 * @param messages the message list used to generate the hash value outputs
	 */
	__global__ void hash_quiescent_location_report_messages(uint* keys, uint* values, xmachine_message_quiescent_location_report_list* messages)
	{
		unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
        glm::vec3 position = glm::vec3(messages->x[index], messages->y[index], messages->z[index]);
		glm::ivec3 grid_position = message_quiescent_location_report_grid_position(position);
		unsigned int hash = message_quiescent_location_report_hash(grid_position);

		keys[index] = hash;
		values[index] = index;
	}

	/** reorder_quiescent_location_report_messages
	 * Reorders the messages accoring to the ordered sort identifiers and builds a Partition Boundary Matrix by looking at the previosu threads sort id.
	 * @param keys the sorted hash keys
	 * @param values the sorted index values
	 * @param matrix the PBM
	 * @param unordered_messages the original unordered message data
	 * @param ordered_messages buffer used to scatter messages into the correct order
	 */
	__global__ void reorder_quiescent_location_report_messages(uint* keys, uint* values, xmachine_message_quiescent_location_report_PBM* matrix, xmachine_message_quiescent_location_report_list* unordered_messages, xmachine_message_quiescent_location_report_list* ordered_messages)
	{
		extern __shared__ int sm_data [];

		int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		//load threads sort key into sm
		uint key = keys[index];
		uint old_pos = values[index];

		sm_data[threadIdx.x] = key;
		__syncthreads();
	
		unsigned int prev_key;

		//if first thread then no prev sm value so get prev from global memory 
		if (threadIdx.x == 0)
		{
			//first thread has no prev value so ignore
			if (index != 0)
				prev_key = keys[index-1];
		}
		//get previous ident from sm
		else	
		{
			prev_key = sm_data[threadIdx.x-1];
		}

		//TODO: Check key is not out of bounds

		//set partition boundaries
		if (index < d_message_quiescent_location_report_count)
		{
			//if first thread then set first partition cell start
			if (index == 0)
			{
				matrix->start[key] = index;
			}

			//if edge of a boundr update start and end of partition
			else if (prev_key != key)
			{
				//set start for key
				matrix->start[key] = index;

				//set end for key -1
				matrix->end_or_count[prev_key] = index;
			}

			//if last thread then set final partition cell end
			if (index == d_message_quiescent_location_report_count-1)
			{
				matrix->end_or_count[key] = index+1;
			}
		}
	
		//finally reorder agent data
		ordered_messages->id[index] = unordered_messages->id[old_pos];
		ordered_messages->x[index] = unordered_messages->x[old_pos];
		ordered_messages->y[index] = unordered_messages->y[old_pos];
		ordered_messages->z[index] = unordered_messages->z[old_pos];
	}

#endif

/** load_next_quiescent_location_report_message
 * Used to load the next message data to shared memory
 * Idea is check the current cell index to see if we can simply get a message from the current cell
 * If we are at the end of the current cell then loop till we find the next cell with messages (this way we ignore cells with no messages)
 * @param messages the message list
 * @param partition_matrix the PBM
 * @param relative_cell the relative partition cell position from the agent position
 * @param cell_index_max the maximum index of the current partition cell
 * @param agent_grid_cell the agents partition cell position
 * @param cell_index the current cell index in agent_grid_cell+relative_cell
 * @return true if a message has been loaded into sm false otherwise
 */
__device__ bool load_next_quiescent_location_report_message(xmachine_message_quiescent_location_report_list* messages, xmachine_message_quiescent_location_report_PBM* partition_matrix, glm::ivec3 relative_cell, int cell_index_max, glm::ivec3 agent_grid_cell, int cell_index)
{
	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];

	int move_cell = true;
	cell_index ++;

	//see if we need to move to a new partition cell
	if(cell_index < cell_index_max)
		move_cell = false;

	while(move_cell)
	{
		//get the next relative grid position 
        if (next_cell3D(&relative_cell))
		{
			//calculate the next cells grid position and hash
			glm::ivec3 next_cell_position = agent_grid_cell + relative_cell;
			int next_cell_hash = message_quiescent_location_report_hash(next_cell_position);
			//use the hash to calculate the start index
			int cell_index_min = tex1Dfetch(tex_xmachine_message_quiescent_location_report_pbm_start, next_cell_hash + d_tex_xmachine_message_quiescent_location_report_pbm_start_offset);
			cell_index_max = tex1Dfetch(tex_xmachine_message_quiescent_location_report_pbm_end_or_count, next_cell_hash + d_tex_xmachine_message_quiescent_location_report_pbm_end_or_count_offset);
			//check for messages in the cell (cell index max is the count for atomic sorting)
#ifdef FAST_ATOMIC_SORTING
			if (cell_index_max > 0)
			{
				//when using fast atomics value represents bin count not last index!
				cell_index_max += cell_index_min; //when using fast atomics value represents bin count not last index!
#else
			if (cell_index_min != 0xffffffff)
			{
#endif
				//start from the cell index min
				cell_index = cell_index_min;
				//exit the loop as we have found a valid cell with message data
				move_cell = false;
			}
		}
		else
		{
			//we have exhausted all the neighbouring cells so there are no more messages
			return false;
		}
	}
	
	//get message data using texture fetch
	xmachine_message_quiescent_location_report temp_message;
	temp_message._relative_cell = relative_cell;
	temp_message._cell_index_max = cell_index_max;
	temp_message._cell_index = cell_index;
	temp_message._agent_grid_cell = agent_grid_cell;

	//Using texture cache
  temp_message.id = tex1Dfetch(tex_xmachine_message_quiescent_location_report_id, cell_index + d_tex_xmachine_message_quiescent_location_report_id_offset); temp_message.x = tex1Dfetch(tex_xmachine_message_quiescent_location_report_x, cell_index + d_tex_xmachine_message_quiescent_location_report_x_offset); temp_message.y = tex1Dfetch(tex_xmachine_message_quiescent_location_report_y, cell_index + d_tex_xmachine_message_quiescent_location_report_y_offset); temp_message.z = tex1Dfetch(tex_xmachine_message_quiescent_location_report_z, cell_index + d_tex_xmachine_message_quiescent_location_report_z_offset); 

	//load it into shared memory (no sync as no sharing between threads)
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_quiescent_location_report));
	xmachine_message_quiescent_location_report* sm_message = ((xmachine_message_quiescent_location_report*)&message_share[message_index]);
	sm_message[0] = temp_message;

	return true;
}


/*
 * get first spatial partitioned quiescent_location_report message (first batch load into shared memory)
 */
__device__ xmachine_message_quiescent_location_report* get_first_quiescent_location_report_message(xmachine_message_quiescent_location_report_list* messages, xmachine_message_quiescent_location_report_PBM* partition_matrix, float x, float y, float z){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];

	// If there are no messages, do not load any messages
	if(d_message_quiescent_location_report_count == 0){
		return nullptr;
	}

	glm::ivec3 relative_cell = glm::ivec3(-2, -1, -1);
	int cell_index_max = 0;
	int cell_index = 0;
	glm::vec3 position = glm::vec3(x, y, z);
	glm::ivec3 agent_grid_cell = message_quiescent_location_report_grid_position(position);
	
	if (load_next_quiescent_location_report_message(messages, partition_matrix, relative_cell, cell_index_max, agent_grid_cell, cell_index))
	{
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_quiescent_location_report));
		return ((xmachine_message_quiescent_location_report*)&message_share[message_index]);
	}
	else
	{
		return nullptr;
	}
}

/*
 * get next spatial partitioned quiescent_location_report message (either from SM or next batch load)
 */
__device__ xmachine_message_quiescent_location_report* get_next_quiescent_location_report_message(xmachine_message_quiescent_location_report* message, xmachine_message_quiescent_location_report_list* messages, xmachine_message_quiescent_location_report_PBM* partition_matrix){
	
	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	// If there are no messages, do not load any messages
	if(d_message_quiescent_location_report_count == 0){
		return nullptr;
	}
	
	if (load_next_quiescent_location_report_message(messages, partition_matrix, message->_relative_cell, message->_cell_index_max, message->_agent_grid_cell, message->_cell_index))
	{
		//get conflict free address of 
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_quiescent_location_report));
		return ((xmachine_message_quiescent_location_report*)&message_share[message_index]);
	}
	else
		return nullptr;
	
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created senescent_location_report message functions */


/** add_senescent_location_report_message
 * Add non partitioned or spatially partitioned senescent_location_report message
 * @param messages xmachine_message_senescent_location_report_list message list to add too
 * @param id agent variable of type int
 * @param x agent variable of type float
 * @param y agent variable of type float
 * @param z agent variable of type float
 */
__device__ void add_senescent_location_report_message(xmachine_message_senescent_location_report_list* messages, int id, float x, float y, float z){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_senescent_location_report_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_senescent_location_report_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_senescent_location_report_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_senescent_location_report Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->id[index] = id;
	messages->x[index] = x;
	messages->y[index] = y;
	messages->z[index] = z;

}

/**
 * Scatter non partitioned or spatially partitioned senescent_location_report message (for optional messages)
 * @param messages scatter_optional_senescent_location_report_messages Sparse xmachine_message_senescent_location_report_list message list
 * @param message_swap temp xmachine_message_senescent_location_report_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_senescent_location_report_messages(xmachine_message_senescent_location_report_list* messages, xmachine_message_senescent_location_report_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_senescent_location_report_count;

		//AoS - xmachine_message_senescent_location_report Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->id[output_index] = messages_swap->id[index];
		messages->x[output_index] = messages_swap->x[index];
		messages->y[output_index] = messages_swap->y[index];
		messages->z[output_index] = messages_swap->z[index];				
	}
}

/** reset_senescent_location_report_swaps
 * Reset non partitioned or spatially partitioned senescent_location_report message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_senescent_location_report_swaps(xmachine_message_senescent_location_report_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

/** message_senescent_location_report_grid_position
 * Calculates the grid cell position given an glm::vec3 vector
 * @param position glm::vec3 vector representing a position
 */
__device__ glm::ivec3 message_senescent_location_report_grid_position(glm::vec3 position)
{
    glm::ivec3 gridPos;
    gridPos.x = floor((position.x - d_message_senescent_location_report_min_bounds.x) * (float)d_message_senescent_location_report_partitionDim.x / (d_message_senescent_location_report_max_bounds.x - d_message_senescent_location_report_min_bounds.x));
    gridPos.y = floor((position.y - d_message_senescent_location_report_min_bounds.y) * (float)d_message_senescent_location_report_partitionDim.y / (d_message_senescent_location_report_max_bounds.y - d_message_senescent_location_report_min_bounds.y));
    gridPos.z = floor((position.z - d_message_senescent_location_report_min_bounds.z) * (float)d_message_senescent_location_report_partitionDim.z / (d_message_senescent_location_report_max_bounds.z - d_message_senescent_location_report_min_bounds.z));

	//do wrapping or bounding
	

    return gridPos;
}

/** message_senescent_location_report_hash
 * Given the grid position in partition space this function calculates a hash value
 * @param gridPos The position in partition space
 */
__device__ unsigned int message_senescent_location_report_hash(glm::ivec3 gridPos)
{
	//cheap bounding without mod (within range +- partition dimension)
	gridPos.x = (gridPos.x<0)? d_message_senescent_location_report_partitionDim.x-1: gridPos.x; 
	gridPos.x = (gridPos.x>=d_message_senescent_location_report_partitionDim.x)? 0 : gridPos.x; 
	gridPos.y = (gridPos.y<0)? d_message_senescent_location_report_partitionDim.y-1 : gridPos.y; 
	gridPos.y = (gridPos.y>=d_message_senescent_location_report_partitionDim.y)? 0 : gridPos.y; 
	gridPos.z = (gridPos.z<0)? d_message_senescent_location_report_partitionDim.z-1: gridPos.z; 
	gridPos.z = (gridPos.z>=d_message_senescent_location_report_partitionDim.z)? 0 : gridPos.z; 

	//unique id
	return ((gridPos.z * d_message_senescent_location_report_partitionDim.y) * d_message_senescent_location_report_partitionDim.x) + (gridPos.y * d_message_senescent_location_report_partitionDim.x) + gridPos.x;
}

#ifdef FAST_ATOMIC_SORTING
	/** hist_senescent_location_report_messages
		 * Kernal function for performing a histogram (count) on each partition bin and saving the hash and index of a message within that bin
		 * @param local_bin_index output index of the message within the calculated bin
		 * @param unsorted_index output bin index (hash) value
		 * @param messages the message list used to generate the hash value outputs
		 * @param agent_count the current number of agents outputting messages
		 */
	__global__ void hist_senescent_location_report_messages(uint* local_bin_index, uint* unsorted_index, int* global_bin_count, xmachine_message_senescent_location_report_list* messages, int agent_count)
	{
		unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		if (index >= agent_count)
			return;
        glm::vec3 position = glm::vec3(messages->x[index], messages->y[index], messages->z[index]);
		glm::ivec3 grid_position = message_senescent_location_report_grid_position(position);
		unsigned int hash = message_senescent_location_report_hash(grid_position);
		unsigned int bin_idx = atomicInc((unsigned int*) &global_bin_count[hash], 0xFFFFFFFF);
		local_bin_index[index] = bin_idx;
		unsorted_index[index] = hash;
	}
	
	/** reorder_senescent_location_report_messages
	 * Reorders the messages accoring to the partition boundary matrix start indices of each bin
	 * @param local_bin_index index of the message within the desired bin
	 * @param unsorted_index bin index (hash) value
	 * @param pbm_start_index the start indices of the partition boundary matrix
	 * @param unordered_messages the original unordered message data
	 * @param ordered_messages buffer used to scatter messages into the correct order
	  @param agent_count the current number of agents outputting messages
	 */
	 __global__ void reorder_senescent_location_report_messages(uint* local_bin_index, uint* unsorted_index, int* pbm_start_index, xmachine_message_senescent_location_report_list* unordered_messages, xmachine_message_senescent_location_report_list* ordered_messages, int agent_count)
	{
		int index = (blockIdx.x *blockDim.x) + threadIdx.x;

		if (index >= agent_count)
			return;

		uint i = unsorted_index[index];
		unsigned int sorted_index = local_bin_index[index] + pbm_start_index[i];

		//finally reorder agent data
		ordered_messages->id[sorted_index] = unordered_messages->id[index];
		ordered_messages->x[sorted_index] = unordered_messages->x[index];
		ordered_messages->y[sorted_index] = unordered_messages->y[index];
		ordered_messages->z[sorted_index] = unordered_messages->z[index];
	}
	 
#else

	/** hash_senescent_location_report_messages
	 * Kernal function for calculating a hash value for each messahe depending on its position
	 * @param keys output for the hash key
	 * @param values output for the index value
	 * @param messages the message list used to generate the hash value outputs
	 */
	__global__ void hash_senescent_location_report_messages(uint* keys, uint* values, xmachine_message_senescent_location_report_list* messages)
	{
		unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
        glm::vec3 position = glm::vec3(messages->x[index], messages->y[index], messages->z[index]);
		glm::ivec3 grid_position = message_senescent_location_report_grid_position(position);
		unsigned int hash = message_senescent_location_report_hash(grid_position);

		keys[index] = hash;
		values[index] = index;
	}

	/** reorder_senescent_location_report_messages
	 * Reorders the messages accoring to the ordered sort identifiers and builds a Partition Boundary Matrix by looking at the previosu threads sort id.
	 * @param keys the sorted hash keys
	 * @param values the sorted index values
	 * @param matrix the PBM
	 * @param unordered_messages the original unordered message data
	 * @param ordered_messages buffer used to scatter messages into the correct order
	 */
	__global__ void reorder_senescent_location_report_messages(uint* keys, uint* values, xmachine_message_senescent_location_report_PBM* matrix, xmachine_message_senescent_location_report_list* unordered_messages, xmachine_message_senescent_location_report_list* ordered_messages)
	{
		extern __shared__ int sm_data [];

		int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		//load threads sort key into sm
		uint key = keys[index];
		uint old_pos = values[index];

		sm_data[threadIdx.x] = key;
		__syncthreads();
	
		unsigned int prev_key;

		//if first thread then no prev sm value so get prev from global memory 
		if (threadIdx.x == 0)
		{
			//first thread has no prev value so ignore
			if (index != 0)
				prev_key = keys[index-1];
		}
		//get previous ident from sm
		else	
		{
			prev_key = sm_data[threadIdx.x-1];
		}

		//TODO: Check key is not out of bounds

		//set partition boundaries
		if (index < d_message_senescent_location_report_count)
		{
			//if first thread then set first partition cell start
			if (index == 0)
			{
				matrix->start[key] = index;
			}

			//if edge of a boundr update start and end of partition
			else if (prev_key != key)
			{
				//set start for key
				matrix->start[key] = index;

				//set end for key -1
				matrix->end_or_count[prev_key] = index;
			}

			//if last thread then set final partition cell end
			if (index == d_message_senescent_location_report_count-1)
			{
				matrix->end_or_count[key] = index+1;
			}
		}
	
		//finally reorder agent data
		ordered_messages->id[index] = unordered_messages->id[old_pos];
		ordered_messages->x[index] = unordered_messages->x[old_pos];
		ordered_messages->y[index] = unordered_messages->y[old_pos];
		ordered_messages->z[index] = unordered_messages->z[old_pos];
	}

#endif

/** load_next_senescent_location_report_message
 * Used to load the next message data to shared memory
 * Idea is check the current cell index to see if we can simply get a message from the current cell
 * If we are at the end of the current cell then loop till we find the next cell with messages (this way we ignore cells with no messages)
 * @param messages the message list
 * @param partition_matrix the PBM
 * @param relative_cell the relative partition cell position from the agent position
 * @param cell_index_max the maximum index of the current partition cell
 * @param agent_grid_cell the agents partition cell position
 * @param cell_index the current cell index in agent_grid_cell+relative_cell
 * @return true if a message has been loaded into sm false otherwise
 */
__device__ bool load_next_senescent_location_report_message(xmachine_message_senescent_location_report_list* messages, xmachine_message_senescent_location_report_PBM* partition_matrix, glm::ivec3 relative_cell, int cell_index_max, glm::ivec3 agent_grid_cell, int cell_index)
{
	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];

	int move_cell = true;
	cell_index ++;

	//see if we need to move to a new partition cell
	if(cell_index < cell_index_max)
		move_cell = false;

	while(move_cell)
	{
		//get the next relative grid position 
        if (next_cell3D(&relative_cell))
		{
			//calculate the next cells grid position and hash
			glm::ivec3 next_cell_position = agent_grid_cell + relative_cell;
			int next_cell_hash = message_senescent_location_report_hash(next_cell_position);
			//use the hash to calculate the start index
			int cell_index_min = tex1Dfetch(tex_xmachine_message_senescent_location_report_pbm_start, next_cell_hash + d_tex_xmachine_message_senescent_location_report_pbm_start_offset);
			cell_index_max = tex1Dfetch(tex_xmachine_message_senescent_location_report_pbm_end_or_count, next_cell_hash + d_tex_xmachine_message_senescent_location_report_pbm_end_or_count_offset);
			//check for messages in the cell (cell index max is the count for atomic sorting)
#ifdef FAST_ATOMIC_SORTING
			if (cell_index_max > 0)
			{
				//when using fast atomics value represents bin count not last index!
				cell_index_max += cell_index_min; //when using fast atomics value represents bin count not last index!
#else
			if (cell_index_min != 0xffffffff)
			{
#endif
				//start from the cell index min
				cell_index = cell_index_min;
				//exit the loop as we have found a valid cell with message data
				move_cell = false;
			}
		}
		else
		{
			//we have exhausted all the neighbouring cells so there are no more messages
			return false;
		}
	}
	
	//get message data using texture fetch
	xmachine_message_senescent_location_report temp_message;
	temp_message._relative_cell = relative_cell;
	temp_message._cell_index_max = cell_index_max;
	temp_message._cell_index = cell_index;
	temp_message._agent_grid_cell = agent_grid_cell;

	//Using texture cache
  temp_message.id = tex1Dfetch(tex_xmachine_message_senescent_location_report_id, cell_index + d_tex_xmachine_message_senescent_location_report_id_offset); temp_message.x = tex1Dfetch(tex_xmachine_message_senescent_location_report_x, cell_index + d_tex_xmachine_message_senescent_location_report_x_offset); temp_message.y = tex1Dfetch(tex_xmachine_message_senescent_location_report_y, cell_index + d_tex_xmachine_message_senescent_location_report_y_offset); temp_message.z = tex1Dfetch(tex_xmachine_message_senescent_location_report_z, cell_index + d_tex_xmachine_message_senescent_location_report_z_offset); 

	//load it into shared memory (no sync as no sharing between threads)
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_senescent_location_report));
	xmachine_message_senescent_location_report* sm_message = ((xmachine_message_senescent_location_report*)&message_share[message_index]);
	sm_message[0] = temp_message;

	return true;
}


/*
 * get first spatial partitioned senescent_location_report message (first batch load into shared memory)
 */
__device__ xmachine_message_senescent_location_report* get_first_senescent_location_report_message(xmachine_message_senescent_location_report_list* messages, xmachine_message_senescent_location_report_PBM* partition_matrix, float x, float y, float z){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];

	// If there are no messages, do not load any messages
	if(d_message_senescent_location_report_count == 0){
		return nullptr;
	}

	glm::ivec3 relative_cell = glm::ivec3(-2, -1, -1);
	int cell_index_max = 0;
	int cell_index = 0;
	glm::vec3 position = glm::vec3(x, y, z);
	glm::ivec3 agent_grid_cell = message_senescent_location_report_grid_position(position);
	
	if (load_next_senescent_location_report_message(messages, partition_matrix, relative_cell, cell_index_max, agent_grid_cell, cell_index))
	{
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_senescent_location_report));
		return ((xmachine_message_senescent_location_report*)&message_share[message_index]);
	}
	else
	{
		return nullptr;
	}
}

/*
 * get next spatial partitioned senescent_location_report message (either from SM or next batch load)
 */
__device__ xmachine_message_senescent_location_report* get_next_senescent_location_report_message(xmachine_message_senescent_location_report* message, xmachine_message_senescent_location_report_list* messages, xmachine_message_senescent_location_report_PBM* partition_matrix){
	
	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	// If there are no messages, do not load any messages
	if(d_message_senescent_location_report_count == 0){
		return nullptr;
	}
	
	if (load_next_senescent_location_report_message(messages, partition_matrix, message->_relative_cell, message->_cell_index_max, message->_agent_grid_cell, message->_cell_index))
	{
		//get conflict free address of 
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_senescent_location_report));
		return ((xmachine_message_senescent_location_report*)&message_share[message_index]);
	}
	else
		return nullptr;
	
}


	
/////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created GPU kernels  */



/**
 *
 */
__global__ void GPUFLAME_TissueTakesDamage(xmachine_memory_TissueBlock_list* agents, RNG_rand48* rand48){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_TissueBlock_count)
        return;
    

	//SoA to AoS - xmachine_memory_TissueTakesDamage Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_TissueBlock agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.damage = agents->damage[index];

	//FLAME function call
	int dead = !TissueTakesDamage(&agent, rand48);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_TissueTakesDamage Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->damage[index] = agent.damage;
}

/**
 *
 */
__global__ void GPUFLAME_TissueSendDamageReport(xmachine_memory_TissueBlock_list* agents, xmachine_message_tissue_damage_report_list* tissue_damage_report_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_TissueBlock_count)
        return;
    

	//SoA to AoS - xmachine_memory_TissueSendDamageReport Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_TissueBlock agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.damage = agents->damage[index];

	//FLAME function call
	int dead = !TissueSendDamageReport(&agent, tissue_damage_report_messages	);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_TissueSendDamageReport Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->damage[index] = agent.damage;
}

/**
 *
 */
__global__ void GPUFLAME_QuiescentMigration(xmachine_memory_Fibroblast_list* agents, xmachine_message_tissue_damage_report_list* tissue_damage_report_messages, xmachine_message_tissue_damage_report_PBM* partition_matrix){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Fibroblast_count)
        return;
    

	//SoA to AoS - xmachine_memory_QuiescentMigration Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];

	//FLAME function call
	int dead = !QuiescentMigration(&agent, tissue_damage_report_messages, partition_matrix);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_QuiescentMigration Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
}

/**
 *
 */
__global__ void GPUFLAME_SenescentMigration(xmachine_memory_Fibroblast_list* agents, xmachine_message_tissue_damage_report_list* tissue_damage_report_messages, xmachine_message_tissue_damage_report_PBM* partition_matrix){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Fibroblast_count)
        return;
    

	//SoA to AoS - xmachine_memory_SenescentMigration Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];

	//FLAME function call
	int dead = !SenescentMigration(&agent, tissue_damage_report_messages, partition_matrix);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_SenescentMigration Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
}

/**
 *
 */
__global__ void GPUFLAME_EarlySenescentMigration(xmachine_memory_Fibroblast_list* agents, xmachine_message_tissue_damage_report_list* tissue_damage_report_messages, xmachine_message_tissue_damage_report_PBM* partition_matrix){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Fibroblast_count)
        return;
    

	//SoA to AoS - xmachine_memory_EarlySenescentMigration Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];

	//FLAME function call
	int dead = !EarlySenescentMigration(&agent, tissue_damage_report_messages, partition_matrix);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_EarlySenescentMigration Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
}

/**
 *
 */
__global__ void GPUFLAME_QuiescentTakesDamage(xmachine_memory_Fibroblast_list* agents, xmachine_message_fibroblast_damage_report_list* fibroblast_damage_report_messages, xmachine_message_fibroblast_damage_report_PBM* partition_matrix, RNG_rand48* rand48){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Fibroblast_count)
        return;
    

	//SoA to AoS - xmachine_memory_QuiescentTakesDamage Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];

	//FLAME function call
	int dead = !QuiescentTakesDamage(&agent, fibroblast_damage_report_messages, partition_matrix, rand48);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_QuiescentTakesDamage Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
}

/**
 *
 */
__global__ void GPUFLAME_QuiescentSendDamageReport(xmachine_memory_Fibroblast_list* agents, xmachine_message_fibroblast_damage_report_list* fibroblast_damage_report_messages, RNG_rand48* rand48){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Fibroblast_count)
        return;
    

	//SoA to AoS - xmachine_memory_QuiescentSendDamageReport Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];

	//FLAME function call
	int dead = !QuiescentSendDamageReport(&agent, fibroblast_damage_report_messages	, rand48);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_QuiescentSendDamageReport Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
}

/**
 *
 */
__global__ void GPUFLAME_Quiescent2Proliferating(xmachine_memory_Fibroblast_list* agents, RNG_rand48* rand48){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Fibroblast_count)
        return;
    

	//SoA to AoS - xmachine_memory_Quiescent2Proliferating Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];

	//FLAME function call
	int dead = !Quiescent2Proliferating(&agent, rand48);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_Quiescent2Proliferating Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
}

/**
 *
 */
__global__ void GPUFLAME_Proliferation(xmachine_memory_Fibroblast_list* agents, xmachine_memory_Fibroblast_list* Fibroblast_agents){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Fibroblast_count)
        return;
    

	//SoA to AoS - xmachine_memory_Proliferation Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];

	//FLAME function call
	int dead = !Proliferation(&agent, Fibroblast_agents);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_Proliferation Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
}

/**
 *
 */
__global__ void GPUFLAME_BystanderEffect(xmachine_memory_Fibroblast_list* agents, xmachine_message_senescent_location_report_list* senescent_location_report_messages, xmachine_message_senescent_location_report_PBM* partition_matrix, RNG_rand48* rand48){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Fibroblast_count)
        return;
    

	//SoA to AoS - xmachine_memory_BystanderEffect Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];

	//FLAME function call
	int dead = !BystanderEffect(&agent, senescent_location_report_messages, partition_matrix, rand48);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_BystanderEffect Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
}

/**
 *
 */
__global__ void GPUFLAME_ExcessiveDamage(xmachine_memory_Fibroblast_list* agents, xmachine_message_fibroblast_damage_report_list* fibroblast_damage_report_messages, xmachine_message_fibroblast_damage_report_PBM* partition_matrix, RNG_rand48* rand48){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Fibroblast_count)
        return;
    

	//SoA to AoS - xmachine_memory_ExcessiveDamage Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];

	//FLAME function call
	int dead = !ExcessiveDamage(&agent, fibroblast_damage_report_messages, partition_matrix, rand48);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_ExcessiveDamage Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
}

/**
 *
 */
__global__ void GPUFLAME_ReplicativeSenescence(xmachine_memory_Fibroblast_list* agents, xmachine_message_doublings_list* doublings_messages, RNG_rand48* rand48){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_ReplicativeSenescence Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    //No partitioned input may launch more threads than required - only load agent data within bounds. 
    if (index < d_xmachine_memory_Fibroblast_count){
    
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];
	} else {
	
	agent.id = 0;
	agent.x = 0;
	agent.y = 0;
	agent.z = 0;
	agent.doublings = 0;
	agent.damage = 0;
	agent.proliferate_bool = 0;
	agent.transition_to_early_sen = 0;
	}

	//FLAME function call
	int dead = !ReplicativeSenescence(&agent, doublings_messages, rand48);
	

	
    //No partitioned input may launch more threads than required - only write agent data within bounds. 
    if (index < d_xmachine_memory_Fibroblast_count){
    //continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_ReplicativeSenescence Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
	}
}

/**
 *
 */
__global__ void GPUFLAME_TransitionToEarlySen(xmachine_memory_Fibroblast_list* agents){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Fibroblast_count)
        return;
    

	//SoA to AoS - xmachine_memory_TransitionToEarlySen Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];

	//FLAME function call
	int dead = !TransitionToEarlySen(&agent);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_TransitionToEarlySen Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
}

/**
 *
 */
__global__ void GPUFLAME_FullSenescence(xmachine_memory_Fibroblast_list* agents, xmachine_message_count_list* count_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_FullSenescence Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    //No partitioned input may launch more threads than required - only load agent data within bounds. 
    if (index < d_xmachine_memory_Fibroblast_count){
    
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];
	} else {
	
	agent.id = 0;
	agent.x = 0;
	agent.y = 0;
	agent.z = 0;
	agent.doublings = 0;
	agent.damage = 0;
	agent.proliferate_bool = 0;
	agent.transition_to_early_sen = 0;
	}

	//FLAME function call
	int dead = !FullSenescence(&agent, count_messages);
	

	
    //No partitioned input may launch more threads than required - only write agent data within bounds. 
    if (index < d_xmachine_memory_Fibroblast_count){
    //continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_FullSenescence Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
	}
}

/**
 *
 */
__global__ void GPUFLAME_ClearanceOfEarlySenescent(xmachine_memory_Fibroblast_list* agents){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Fibroblast_count)
        return;
    

	//SoA to AoS - xmachine_memory_ClearanceOfEarlySenescent Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];

	//FLAME function call
	int dead = !ClearanceOfEarlySenescent(&agent);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_ClearanceOfEarlySenescent Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
}

/**
 *
 */
__global__ void GPUFLAME_ClearanceOfSenescent(xmachine_memory_Fibroblast_list* agents){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Fibroblast_count)
        return;
    

	//SoA to AoS - xmachine_memory_ClearanceOfSenescent Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];

	//FLAME function call
	int dead = !ClearanceOfSenescent(&agent);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_ClearanceOfSenescent Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
}

/**
 *
 */
__global__ void GPUFLAME_DetectDamage(xmachine_memory_Fibroblast_list* agents, xmachine_message_tissue_damage_report_list* tissue_damage_report_messages, xmachine_message_tissue_damage_report_PBM* partition_matrix){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Fibroblast_count)
        return;
    

	//SoA to AoS - xmachine_memory_DetectDamage Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];

	//FLAME function call
	int dead = !DetectDamage(&agent, tissue_damage_report_messages, partition_matrix);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_DetectDamage Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
}

/**
 *
 */
__global__ void GPUFLAME_RepairDamage(xmachine_memory_Fibroblast_list* agents){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Fibroblast_count)
        return;
    

	//SoA to AoS - xmachine_memory_RepairDamage Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];

	//FLAME function call
	int dead = !RepairDamage(&agent);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_RepairDamage Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
}

/**
 *
 */
__global__ void GPUFLAME_DamageRepaired(xmachine_memory_Fibroblast_list* agents){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Fibroblast_count)
        return;
    

	//SoA to AoS - xmachine_memory_DamageRepaired Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Fibroblast agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.z = agents->z[index];
	agent.doublings = agents->doublings[index];
	agent.damage = agents->damage[index];
	agent.proliferate_bool = agents->proliferate_bool[index];
	agent.transition_to_early_sen = agents->transition_to_early_sen[index];

	//FLAME function call
	int dead = !DamageRepaired(&agent);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_DamageRepaired Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->z[index] = agent.z;
	agents->doublings[index] = agent.doublings;
	agents->damage[index] = agent.damage;
	agents->proliferate_bool[index] = agent.proliferate_bool;
	agents->transition_to_early_sen[index] = agent.transition_to_early_sen;
}

	
	
/* Graph utility functions */



/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Rand48 functions */

__device__ static glm::uvec2 RNG_rand48_iterate_single(glm::uvec2 Xn, glm::uvec2 A, glm::uvec2 C)
{
	unsigned int R0, R1;

	// low 24-bit multiplication
	const unsigned int lo00 = __umul24(Xn.x, A.x);
	const unsigned int hi00 = __umulhi(Xn.x, A.x);

	// 24bit distribution of 32bit multiplication results
	R0 = (lo00 & 0xFFFFFF);
	R1 = (lo00 >> 24) | (hi00 << 8);

	R0 += C.x; R1 += C.y;

	// transfer overflows
	R1 += (R0 >> 24);
	R0 &= 0xFFFFFF;

	// cross-terms, low/hi 24-bit multiplication
	R1 += __umul24(Xn.y, A.x);
	R1 += __umul24(Xn.x, A.y);

	R1 &= 0xFFFFFF;

	return glm::uvec2(R0, R1);
}

//Templated function
template <int AGENT_TYPE>
__device__ float rnd(RNG_rand48* rand48){

	int index;
	
	//calculate the agents index in global agent list
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x * gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y * width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	glm::uvec2 state = rand48->seeds[index];
	glm::uvec2 A = rand48->A;
	glm::uvec2 C = rand48->C;

	int rand = ( state.x >> 17 ) | ( state.y << 7);

	// this actually iterates the RNG
	state = RNG_rand48_iterate_single(state, A, C);

	rand48->seeds[index] = state;

	return (float)rand/2147483647;
}

__device__ float rnd(RNG_rand48* rand48){
	return rnd<DISCRETE_2D>(rand48);
}

#endif //_FLAMEGPU_KERNELS_H_
